#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__device__ void printResult(unsigned char result) {
    if (result == 16) {
        printf("Result - void\n");
        return;
    }
    printf("Result - Ply: (%d, %d), Val: %d\n", (result >> 4) & 7, result & 15, (result >> 7) == 0 ? -1 : 1);
}

__device__ void printResultArray(unsigned char* resultArray, unsigned int level) {
    for (int i = 0; i < level; i++) {
        printf("   ");
    }
    if (!resultArray) {
        printf("ResultArray - NULL\n");
        return;
    }
    unsigned char result = resultArray[0];
    if (result == 16) {
        printf("ResultArray - void\n");
        return;
    }
    printf("ResultArray - [\n");
    unsigned int index = 0;
    do {
        printf("   ");
        for (int j = 0; j < level; j++) {
            printf("   ");
        }
        printResult(result);
        index++;
        result = resultArray[index];
    } while (result != 16);
    for (int i = 0; i < level; i++) {
        printf("   ");
    }
    printf("]\n");
}

__device__ void printEntry(StackEntry* entry, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return;
    }
    if (!entry) {
        printf("StackEntry - NULL\n");
        return;
    }
    printf("StackEntry - {\n");
    printf("   ");
    printNim(entry->board, numRows);
    printf("   Alpha: %d, Beta: %d, Player: %d\n", entry->alpha, entry->beta, entry->player);
    printf("   Depth: %d, PlyIndex: %d, StackIndex: %d\n", entry->depth, entry->plyIndex, entry->stackIndex);
    printResultArray(entry->evaluations, 1);
    printf("   ");
    printResult(entry->result);
    printf("} \n");
}

__device__ unsigned char minResultArray(unsigned char results[]) {
    unsigned char result = results[0];
    if (result == 16) {
        printf("Empty resultArray!\n");
        return 16;
    }

    unsigned int min_index = 0;
    int min_val = 2;
    unsigned int index = 0;
    int val;
    do {
        val = result >> 7 == 0 ? -1 : 1;
        if(val < min_val) {
            min_index = index;
            min_val = val;
        }
        index++;
        result = results[index];
    } while (result != 16);
    return results[min_index];
}

__device__ unsigned char maxResultArray(unsigned char results[]) {
    unsigned char result = results[0];
    if (result == 16) {
        printf("Empty resultArray!\n");
        return 16;
    }

    unsigned int max_index = 0;
    int max_val = -2;
    unsigned int index = 0;
    int val;
    do {
        val = result >> 7 == 0 ? -1 : 1;
        if(val > max_val) {
            max_index = index;
            max_val = val;
        }
        index++;
        result = results[index];
    } while (result != 16);
    return results[max_index];
}

__device__ void stackPush(Stack* stack, unsigned int maxStackSize, unsigned int board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, unsigned char evaluations[], unsigned char result) {
    if (stack->stackSize == maxStackSize) {
        printf("the stack size is not enough!\n");
        return;
    }
    unsigned int index = stack->stackSize;
    stack->array[index].board = board;
    stack->array[index].alpha = alpha;
    stack->array[index].beta = beta;
    stack->array[index].player = player;
    stack->array[index].depth = depth;
    stack->array[index].plyIndex = plyIndex;
    stack->array[index].stackIndex = stackIndex;
    stack->array[index].evaluations = evaluations;
    stack->array[index].result = result;
    stack->stackSize++;
}

__device__ void stackPop(Stack* stack, StackEntry* entry) {
    if (stack->stackSize < 1) {
        printf("the stack is empty!\n");
        return;
    }
    stack->stackSize--;
    unsigned int index = stack->stackSize;
    entry->board = stack->array[index].board;
    entry->alpha = stack->array[index].alpha;
    entry->beta = stack->array[index].beta;
    entry->player = stack->array[index].player;
    entry->depth = stack->array[index].depth;
    entry->plyIndex = stack->array[index].plyIndex;
    entry->stackIndex = stack->array[index].stackIndex;
    entry->evaluations = stack->array[index].evaluations;
    entry->result = stack->array[index].result;
}
