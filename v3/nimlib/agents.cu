#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include "nimlib.cuh"

__global__ void GPU_minmax(unsigned int nim, unsigned int numRows, unsigned char* moves, unsigned char numPlys, unsigned char* results) {
    const unsigned int maxMoves = 26; // + 1 for the ending code (16)

    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned int stopComputation = 0;
    // stopComputation values:
    // 0 - Keep calculating
    // 1 - Calculate only the global result
    // 2 - Calculate the shared and the global result

    if (bid >= numPlys)
        return;

    // const unsigned int totArrays = maxMoves + (maxMoves - 1) * (maxMoves - 1); // 1 + n + n^2
    const unsigned int totArrays = maxMoves; // 1 + n

    __shared__ unsigned char sharedResults[totArrays][maxMoves];
    __shared__ unsigned char sharedMoves[totArrays][maxMoves];
    __shared__ unsigned char sharedNumPlys[totArrays];
    __shared__ unsigned int sharedBoards[totArrays];

    /* Level 0 */
    // player = 1;

    if (tid == 0) {
        // calculate the new board and invert the current player
        // select the move from bid
        // calculate the resulting board for the current move
        unsigned char move = moves[bid];
        unsigned int newBoard = nimming(nim, numRows, move);
        // player = -1;

        // check if the game is ended, if yes update the results
        if (!isNotEnded(newBoard)) {
            // 0 -> -1
            // 1 -> 1
            results[bid] = 0 + (move & 127);

            sharedNumPlys[0] = 0;

            // if bid == 0 and tid == 0 => should put 16 in the last element
            if (bid == 0)
                stopComputation = 1;
        }

        // calculate the new moves on shared array
        sharedBoards[0] = newBoard;
        sharedNumPlys[0] = possibleMoves(newBoard, numRows, sharedMoves[0], -1);
    }

    __syncthreads();

    /* Level 1 */
    // player = -1;
    unsigned int l1_index;
    unsigned int shift;
    if (tid < maxMoves-1) {
        l1_index = tid + 1;
        shift = 0;
    } else {
        l1_index = (tid - (maxMoves - 1)) / (maxMoves - 2) + 1;
        shift = (tid - (maxMoves - 1)) % (maxMoves - 2) + 1;
    }

    __syncthreads();

    if (stopComputation == 0 && tid < sharedNumPlys[0]) {
        // apply tid move
        unsigned char move = sharedMoves[0][tid];
        unsigned int newBoard = nimming(sharedBoards[0], numRows, move);
        // player = 1;

        // check if nim is ended
        if (!isNotEnded(newBoard)) {
            sharedResults[0][tid] = 128 + (move & 127);
            sharedNumPlys[l1_index] = 0;

            // jump to min/max evaluation            
            if (tid == 0)
                stopComputation = 2;
        }

        sharedBoards[l1_index] = newBoard;
        sharedNumPlys[l1_index] = possibleMoves(newBoard, numRows, sharedMoves[l1_index], -1);
    }

    __syncthreads();

    /* Level 2 */
    // player = 1;

    // unsigned int l2_index = tid + 1 + maxMoves-1; // from 26 to 651
    // if (bid == 1) printf("T %d - SH %d - L1 %d - Num %d\n", tid, shift, l1_index, sharedNumPlys[l1_index]);
    unsigned int newBoard = 0;
    if (stopComputation == 0 && sharedNumPlys[l1_index] > shift) {
        unsigned char move = sharedMoves[l1_index][shift];
        newBoard = nimming(sharedBoards[l1_index], numRows, move);
        // player = -1;

        // check if nim is ended
        if (!isNotEnded(newBoard)) {
            sharedResults[l1_index][shift] = 128 + (move & 127);
            // sharedNumPlys[l2_index] = 0;

            // jump to min/max evaluation            
            if (tid == 0)
                stopComputation = 3;
        }
        // sharedNumPlys[l2_index] = possibleMoves(newBoard, numRows, sharedMoves[l2_index], -1);
        // sharedBoards[l2_index] = newBoard;
    }

    /* Level 3 */
    // player = -1;
    // Can be done in global memory [we don't have enough threads]
    
    __syncthreads();

    if (stopComputation == 0 && newBoard != 0) {
        // start to calculate the minmax, store the result in sharedResults
        standard_minmax(newBoard, numRows, -1, shift, sharedResults[l1_index]);
    }

    if (tid >= sharedNumPlys[0])
        return;

    if (stopComputation != 2) {
        // calculate the best move from the shared results
        sharedResults[l1_index][sharedNumPlys[l1_index]] = 16;
        sharedResults[0][shift] = minResultArray(sharedResults[l1_index]);
    }

    __syncthreads();
    
    if (tid != 0)
        return;

    if (stopComputation != 1) {
        // calculate the best move from the shared results
        sharedResults[0][sharedNumPlys[0]] = 16;
        results[bid] = maxResultArray(sharedResults[0]);
    }

    __syncthreads();

    if (bid != 0) return;

    // insert the termination char
    results[numPlys] = 16;
}

// sharedResults is the output
__device__ void standard_minmax(unsigned int nim, unsigned int numRows, int player, unsigned int tid, unsigned char* sharedResults) {
    const unsigned int maxMoves = 26;
    const unsigned int maxDepth = 5;
    const unsigned int maxStackSize = 8;
    /*
    | Max Depth | Max Stack Size |
    | --------- | -------------- |
    | 1         | 4              |
    | 2         | 5              |
    | 3         | 6              |
    | 4         | 7              |
    | 5         | 8              |
    | 6         | 9              |
    | 7         | 10             |
    */

    // init the stack
    Stack stack;
    stack.stackSize = 0;
    StackEntry entries[maxStackSize];
    stack.array = entries;
    
    // push the very first empty entry
    stackPush(&stack, maxStackSize, 0, 0, 0, 0, 0, 0, 0, NULL, 16);

    // push the first meaningful entry
    unsigned char evaluations[maxStackSize-1][maxMoves];
    evaluations[0][0] = 16;
    stackPush(&stack, maxStackSize, nim, -1, 1, 1, 0, -1, stack.stackSize-1, evaluations[0], 16);

    StackEntry entry;
    
    // while there are moves to evaluate
    while (stack.stackSize > 1) {
        __syncthreads();

        stackPop(&stack, &entry);

        // stop if the max depth was reached
        if (entry.depth > maxDepth) {
            stack.array[entry.stackIndex].result = entry.player == -1 ? 128 : 0;
            continue;
        }
        // stop if the game ended
        if (!isNotEnded(entry.board)) {
            stack.array[entry.stackIndex].result = entry.player == -1 ? 0 : 128;
            continue;
        }
        // calculate the posible moves
        unsigned char curr_move = possibleMoves(entry.board, numRows, NULL, entry.plyIndex+1);
        __syncthreads();
        // use the calculated result if it's not the first move
        if (entry.plyIndex != -1) {
            unsigned char prev_move = possibleMoves(entry.board, numRows, NULL, entry.plyIndex);
            // exploit the previous result calculation
            entry.evaluations[entry.plyIndex] = (entry.result & 128) + prev_move;
            entry.evaluations[entry.plyIndex + 1] = 16;
            int val = entry.result >> 7 == 0 ? -1 : 1;
            // update alpha or beta
            if (entry.player == 1) {
                if (entry.beta > val) entry.beta = val;
            } else {
                if (entry.alpha < val) entry.alpha = val;
            }
            // stop if it's the last move or it's time to prune
            if (curr_move == 16 || entry.beta <= entry.alpha) {
                unsigned char r;
                if (entry.player == 1) {
                    r = minResultArray(entry.evaluations);
                } else {
                    r = maxResultArray(entry.evaluations);
                }
                stack.array[entry.stackIndex].result = r;
                continue;
            }
        }
        __syncthreads();
        // evaluate the next move
        unsigned int newBoard;
        newBoard = nimming(entry.board, numRows, curr_move);
        // push the previous state
        stackPush(&stack, maxStackSize, entry.board, entry.alpha, entry.beta, entry.player, entry.depth, entry.plyIndex + 1, entry.stackIndex, entry.evaluations, entry.result);
        __syncthreads();
        // push the current state (after making the move)
        evaluations[stack.stackSize-1][0] = 16;
        stackPush(&stack, maxStackSize, newBoard, entry.alpha, entry.beta, -(entry.player), entry.depth + 1, -1, stack.stackSize - 1, evaluations[stack.stackSize-1], 16);
    }
    stackPop(&stack, &entry);
    // push the result into the shared results
    sharedResults[tid] = entry.result;
}

unsigned int randomStrategy(unsigned int nim, unsigned int numRows, bool print) {
    unsigned int maxMoves = numRows * numRows + 1;
    unsigned char moves[maxMoves];
    moves[0] = 16;
    unsigned char numMoves = possibleMoves(nim, numRows, moves, -1);

    if (numMoves == 0) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % numMoves;
    unsigned char ply = moves[r];
    nim = nimming(nim, numRows, ply);
    if (print){
        printf("Random - (%d, %d)\n", (ply >> 4) & 7, ply & 15);
        printNim(nim, numRows);
        printf("\n");
    } 

    return nim;
}
