#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nim.cuh"

__host__ __device__ void printNimply(unsigned char nimply) {
    if (!nimply) {
        printf("Nimply - NULL");
        return;
    }
    printf("Nimply - Row: %d, Num: %d\n", (nimply >> 4) & 7, nimply & 15);
}

__host__ __device__ void printNim(unsigned int nim, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return;
    }
    printf("Nim - %d", nim & 1);
    unsigned int shift = 0;
    for (unsigned int i = 1; i < numRows; i++) {
        unsigned int mask;
        switch(i) {
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        shift += 4;
        printf(", %d", (nim >> shift) & mask);
    }
    printf("\n");
}

__host__ __device__ void printMovesArray(unsigned char movesArray[]) {
    unsigned char move = movesArray[0];
    if (move == 16) {
        printf("MovesArray - void\n");
        return;
    }
    printf("MovesArray - [\n");
    unsigned int index = 0;
    do {
        printf("   ");
        printNimply(move);
        index++;
        move = movesArray[index];
    } while (move != 16);
    printf("]\n");
}

unsigned int createNim(unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return 0;
    }
    unsigned int nim = 0;
    unsigned int shift = 0;
    for (unsigned int i = 0; i < numRows; i++) {
        unsigned int num = i * 2 + 1;
        nim += (num << shift);
        shift += 4;
    }
    return nim;
}

__host__ __device__ bool isNotEnded(unsigned int nim) {
    return nim != 0;
}

__host__ __device__ unsigned int nimming(unsigned int nim, unsigned int numRows, unsigned char nimply) {
    unsigned int row = (nimply >> 4) & 7;
    unsigned int numSticks = nimply & 15;
    if (numSticks < 1) {
        printf("Not a valid move!\n");
        return nim;
    }
    if (numRows <= row) {
        printf("Not enougth rows!\n");
        return nim;
    }

    unsigned int mask;
    switch(row) {
        case 0:
            mask = 1;
            break;
        case 1:
            mask = 3;
            break;
        case 2:
            mask = 7;
            break;
        case 3:
            mask = 7;
            break;
        default:
            mask = 15;
    }
    unsigned int shift = 4 * row;

    unsigned int sticks = (nim >> shift) & mask;
    if (sticks < numSticks) {
        printf("Not enougth sticks!\n");
        return nim;
    }
    unsigned int newMask = 4294967295 ^ (15 << shift);
    sticks = (sticks - numSticks) << shift;
    return (nim & newMask) | sticks;
}

__host__ __device__ unsigned char possibleMoves(unsigned int nim, unsigned int numRows, unsigned char* output, int index) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return 0;
    }
    unsigned int shift = 0;
    int i = 0;
    for (unsigned char r = 0; r < numRows; r++) {
        unsigned int mask;
        switch(r) {
            case 0: // 1
                mask = 1;
                break;
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        unsigned int c = (nim >> shift) & mask;
        shift += 4;
        for (unsigned char o = 1; o <= c; o++) {
            if (index == -1) {
                output[i] = (r << 4) + o;
            } else if (index == i) {
                return (r << 4) + o;
            }
            i++;
        }
    }
    if (index >= i) {
        if (index > i) printf("Index: %d - I: %d\n", index, i);
        return 16;
    }

    output[i] = 16; // 1 << 4 - row 1, num 0
    return i;
}

__host__ __device__ unsigned int nim_sum(unsigned int nim, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return false;
    }
    unsigned int nim_sum = nim & 1;
    unsigned int shift = 0;
    for (unsigned int i = 1; i < numRows; i++) {
        unsigned int mask;
        switch(i) {
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        shift += 4;
        nim_sum ^= ((nim >> shift) & mask);
    }
    return nim_sum; 
}
