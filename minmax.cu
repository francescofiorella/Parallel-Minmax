#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nim.h"
#include "./nimlib/agents.h"
#include "./nimlib/utils.h"
#include "nim.cuh"

__global__ void GPU_minmax(Nim* nim, unsigned int* rows, Nimply* ply) {
    // Associate rows to nim
    nim->rows = rows;

    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    ResultArray* results;
    MovesArray* moves;

    unsigned int maxMoves = nim->numRows * nim->numRows;

    // resulve bug of global variables

    if (bid == 0 && tid == 0) {
        // initialize the global output
        // the max number of results is equal to the available moves => the max num of moves is rows^2
        results = GPU_createResultArray(maxMoves);
        
        // calculate the first moves
        moves = GPU_possibleMoves(nim);
    }

    __syncthreads();

    if (bid >= moves->numItems) {
        return;
    }

    __syncthreads();

    if (tid == 0) {
        // calculate the new board and invert the current player
        Nim* newBoard = GPU_deepcopyNim(nim);
        // select the move from bid
        // calculate the resulting board for the current move
        GPU_nimming(newBoard, moves->array[bid]);

        // check if the game is ended

        // create the shared stack (maybe initially use another global stack)
        unsigned int maxStackSize = 100000; /* TODO */
        // the max number of evaluations is equal to the available moves => the max num of moves is rows^2
        Stack* stack;
        StackEntry* entry;
        stack = GPU_createStack(maxStackSize);
        // entry = GPU_createStackEntry(NULL, 0, 0, 0, 0, NULL, NULL);
        // GPU_stackPush(stack, entry);
        entry = GPU_createStackEntry(newBoard, -1, 0, -1, stack->stackSize-1, createResultArray(maxMoves), NULL);
        GPU_stackPush(stack, entry);
    }

    __syncthreads();

    // start to calculate, one move for each thread

    // when all secondary threads finished
    __syncthreads();

    // calculate the best move from the global results
    ply = GPU_minResultArray(entry->evaluations)->ply;

    // free global results
    GPU_destroyMovesArray(results); // to keep ply

    printf("GPU nimply:\n");
    GPU_printNimply(ply);
}

// let's remove alpha beta pruning and max depth constrains
// push in the stack every move, using the same evaluations pointer
// at every move, push every move below
// use the depth value to discriminate between layers
// until 1024 (!?)
