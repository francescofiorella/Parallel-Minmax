#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
// #include "./nimlib/nimlib.h"
#include "./nimlib_GPU/nimlib.cuh"

int main_CPU(void);
int main_GPU(void);

#define NUM_ROWS 5

int main(void) {
    return main_GPU();
}

int main_GPU(void) {
    // remember to include ONLY the GPU library [nimlib_GPU/nimlib.cuh]

    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    Nim* nim; // nim on the host CPU machine
    Nim* dev_nim; // nim for the GPU device
    unsigned int* dev_rows;
    Nimply* move; // the move on the host CPU machine
    Nimply* dev_move; // the move for the GPU device

    ResultArray* results;
    ResultArray* dev_results;
    Result* dev_resultArray;
    MovesArray* moves;
    MovesArray* dev_moves;
    Nimply* dev_plys;

    // Allocate the memory on the CPU, initialize nim
    nim = (Nim*)malloc(sizeof(Nim));
    unsigned int rows[NUM_ROWS];
    createNim(nim, rows, NUM_ROWS);
    printRows(nim);

    results = (ResultArray*)malloc(sizeof(ResultArray));
    results->numItems = 0;
    results->array = (Result*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Result));
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    moves->numItems = 0;
    moves->array = (Nimply*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Nimply));

    int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(nim) && a == 0) {
        a++;

        // Allocate the memory on the GPU
        hipMalloc( (void**)&dev_nim, sizeof(Nim) );
        hipMalloc( (void**)&dev_rows, NUM_ROWS * sizeof(unsigned int) );
        hipMalloc( (void**)&dev_move, sizeof(Nimply) );

        hipMalloc( (void**)&dev_results, sizeof(ResultArray) );
        hipMalloc( (void**)&dev_resultArray, NUM_ROWS*NUM_ROWS * sizeof(Result) );
        hipMalloc( (void**)&dev_moves, sizeof(MovesArray) );
        hipMalloc( (void**)&dev_plys, NUM_ROWS*NUM_ROWS * sizeof(Nimply) );

        // Copy nim to the GPU
        hipMemcpy( dev_nim, nim, sizeof(Nim), hipMemcpyHostToDevice );
        hipMemcpy( dev_rows, nim->rows, NUM_ROWS * sizeof(unsigned int), hipMemcpyHostToDevice );

        hipMemcpy( dev_results, results, sizeof(ResultArray), hipMemcpyHostToDevice );
        hipMemcpy( dev_resultArray, results->array, NUM_ROWS*NUM_ROWS * sizeof(Result), hipMemcpyHostToDevice );
        hipMemcpy( dev_moves, moves, sizeof(MovesArray), hipMemcpyHostToDevice );
        hipMemcpy( dev_plys, moves->array, NUM_ROWS*NUM_ROWS * sizeof(Nimply), hipMemcpyHostToDevice );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(dev_nim, dev_rows, dev_results, dev_resultArray, dev_moves, dev_plys, dev_move);

        // Copy the move back from the GPU to the CPU
        hipMemcpy( move, dev_move, sizeof(Nimply), hipMemcpyDeviceToHost );

        // Free the memory allocated on the GPU
        hipFree( dev_rows );
        hipFree( dev_nim );
        hipFree( dev_move );

        nimming(nim, move);

        // Free the memory we allocated on the CPU
        free(move);

        printf("GPU Minmax: ");
        printRows(nim);

        // The CPU perform a random move
        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}

/* int main_CPU(void) {
    // remember to include ONLY the CPU library [nimlib/nimlib.h]

    Nim* nim = createNim(NUM_ROWS);
    printRows(nim);

    
    Nimply* move;
    while(isNotEnded(nim)) {
        move = minmax(nim);
        nimming(nim, move);
        destroyNimply(move);
        printf("Minmax: ");
        printRows(nim);

        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
} */
