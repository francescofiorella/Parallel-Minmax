#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nim.h"
#include "./nimlib/agents.h"
#include "nim.cuh"

int main_CPU(void);
int main_GPU(void);

#define NUM_ROWS 5

int main(void) {
    return main_GPU();
}

int main_GPU(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(1024);

    // Creation of the memory pointers
    Nim* nim; // nim on the host CPU machine
    Nim* dev_nim; // nim for the GPU device
    unsigned int* dev_rows;
    Nimply* move; // the move on the host CPU machine
    Nimply* dev_move; // the move for the GPU device

    // Allocate the memory on the CPU, initialize nim
    nim = createNim(NUM_ROWS);
    printRows(nim);

    int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(nim) && a == 0) {
        a++;

        // Allocate the memory on the CPU, initialize nimply
        move = createNimply(0, 0);

        // Allocate the memory on the GPU
        hipMalloc( (void**)&dev_nim, sizeof(Nim) );
        hipMalloc( (void**)&dev_rows, NUM_ROWS * sizeof(unsigned int) );
        hipMalloc( (void**)&dev_move, sizeof(Nimply) );

        // Copy nim to the GPU
        hipMemcpy( dev_nim, nim, sizeof(Nim), hipMemcpyHostToDevice );
        // hipMemcpy( dev_nim->rows, nim->rows, NUM_ROWS * sizeof(unsigned int), hipMemcpyHostToDevice );
        hipMemcpy( dev_rows, nim->rows, NUM_ROWS * sizeof(unsigned int), hipMemcpyHostToDevice );
        hipMemcpy( dev_move, move, sizeof(Nimply), hipMemcpyHostToDevice );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(dev_nim, dev_rows, dev_move);

        // Copy the move back from the GPU to the CPU
        hipMemcpy( move, dev_move, sizeof(Nimply), hipMemcpyDeviceToHost );

        // Free the memory allocated on the GPU
        hipFree( dev_rows );
        hipFree( dev_nim );
        hipFree( dev_move );

        nimming(nim, move);

        // Free the memory we allocated on the CPU
        destroyNimply(move);

        printf("GPU Minmax: ");
        printRows(nim);

        // The CPU perform a random move
        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}

int main_CPU(void) {
    Nim* nim = createNim(NUM_ROWS);
    printRows(nim);

    
    Nimply* move;
    while(isNotEnded(nim)) {
        move = minmax(nim);
        nimming(nim, move);
        destroyNimply(move);
        printf("Minmax: ");
        printRows(nim);

        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}