#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/GPU/nimlib.cuh"

#define NUM_ROWS 5

#define gpuErrchk(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    Nim* nim; // nim on the host CPU machine
    Nim* dev_nim; // nim for the GPU device
    unsigned int* dev_rows;
    Nimply* move; // the move on the host CPU machine
    Nimply* dev_move; // the move for the GPU device

    ResultArray* results;
    ResultArray* dev_results;
    Result* dev_resultArray;
    MovesArray* moves;
    MovesArray* dev_moves;
    Nimply* dev_plys;

    // Allocate the memory on the CPU, initialize nim
    nim = (Nim*)malloc(sizeof(Nim));
    if (!nim) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    unsigned int rows[NUM_ROWS];
    createNim(nim, rows, NUM_ROWS);
    printf("\n");
    printf("Initial board:\n");
    printRows(nim);
    printf("\n");

    results = (ResultArray*)malloc(sizeof(ResultArray));
    if (!results) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    results->numItems = 0;
    results->array = (Result*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Result));
    if (!results->array) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    if (!moves) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    moves->numItems = 0;
    moves->array = (Nimply*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Nimply));
    if (!moves->array) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(nim) && a == 0) {
        a++;

        // calculate the first level of the tree
        possibleMoves(nim, moves);
        results->numItems = moves->numItems;

        // Allocate the memory on the GPU
        gpuErrchk( hipMalloc( (void**)&dev_nim, sizeof(Nim) ) );
        gpuErrchk( hipMalloc( (void**)&dev_rows, NUM_ROWS * sizeof(unsigned int) ) );
        gpuErrchk( hipMalloc( (void**)&dev_move, sizeof(Nimply) ) );

        gpuErrchk( hipMalloc( (void**)&dev_results, sizeof(ResultArray) ) );
        gpuErrchk( hipMalloc( (void**)&dev_resultArray, NUM_ROWS*NUM_ROWS * sizeof(Result) ) );
        gpuErrchk( hipMalloc( (void**)&dev_moves, sizeof(MovesArray) ) );
        gpuErrchk( hipMalloc( (void**)&dev_plys, NUM_ROWS*NUM_ROWS * sizeof(Nimply) ) );

        // Copy nim to the GPU
        gpuErrchk( hipMemcpy( dev_nim, nim, sizeof(Nim), hipMemcpyHostToDevice ) );
        gpuErrchk( hipMemcpy( dev_rows, nim->rows, NUM_ROWS * sizeof(unsigned int), hipMemcpyHostToDevice ) );

        gpuErrchk( hipMemcpy( dev_results, results, sizeof(ResultArray), hipMemcpyHostToDevice ) );
        gpuErrchk( hipMemcpy( dev_resultArray, results->array, NUM_ROWS*NUM_ROWS * sizeof(Result), hipMemcpyHostToDevice ) );
        gpuErrchk( hipMemcpy( dev_moves, moves, sizeof(MovesArray), hipMemcpyHostToDevice ) );
        gpuErrchk( hipMemcpy( dev_plys, moves->array, NUM_ROWS*NUM_ROWS * sizeof(Nimply), hipMemcpyHostToDevice ) );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(dev_nim, dev_rows, dev_results, dev_resultArray, dev_moves, dev_plys, dev_move);
        
        gpuErrchk( hipPeekAtLastError() );

        // Copy the move back from the GPU to the CPU
        gpuErrchk( hipMemcpy( move, dev_move, sizeof(Nimply), hipMemcpyDeviceToHost ) );

        // Free the memory allocated on the GPU
        hipFree( dev_nim );
        hipFree( dev_rows );
        hipFree( dev_results );
        hipFree( dev_resultArray );
        hipFree( dev_moves );
        hipFree( dev_plys );
        hipFree( dev_move );

        nimming(nim, move);

        // Free the memory we allocated on the CPU
        free(move);
        free(results->array);
        free(results);
        free(moves->array);
        free(moves);

        printf("GPU Minmax:\n");
        printRows(nim);
        printf("\n");

        // The CPU perform a random move
        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random:\n");
            printRows(nim);
            printf("\n");
        }
    }
    
    destroyNim(nim);
    return 0;
}
