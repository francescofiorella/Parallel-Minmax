#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nim.h"
#include "./nimlib/agents.h"

#define NUM_ROWS 5

int main(void) {
    Nim* nim = createNim(NUM_ROWS);
    printRows(nim);

    
    Nimply* move;
    while(isNotEnded(nim)) {
        move = minmax(nim);
        nimming(nim, move);
        destroyNimply(move);
        printf("Minmax: ");
        printRows(nim);

        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}
