#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
// #include "./nimlib/nimlib.h"
#include "./nimlib_GPU/nimlib.cuh"

int main_CPU(void);
int main_GPU(void);

#define NUM_ROWS 5

int main(void) {
    return main_GPU();
}

int main_GPU(void) {
    // remember to include ONLY the GPU library [nimlib_GPU/nimlib.cuh]

    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    Nim* nim; // nim on the host CPU machine
    Nim* dev_nim; // nim for the GPU device
    unsigned int* dev_rows;
    Nimply* move; // the move on the host CPU machine
    Nimply* dev_move; // the move for the GPU device

    // Allocate the memory on the CPU, initialize nim
    nim = (Nim*)malloc(sizeof(Nim));
    nim->rows = (unsigned int*)malloc(NUM_ROWS * sizeof(unsigned int));
    createNim(nim, NUM_ROWS);
    printRows(nim);

    int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(nim) && a == 0) {
        a++;

        // Allocate the memory on the GPU
        hipMalloc( (void**)&dev_nim, sizeof(Nim) );
        hipMalloc( (void**)&dev_rows, NUM_ROWS * sizeof(unsigned int) );
        hipMalloc( (void**)&dev_move, sizeof(Nimply) );

        // Copy nim to the GPU
        hipMemcpy( dev_nim, nim, sizeof(Nim), hipMemcpyHostToDevice );
        hipMemcpy( dev_rows, nim->rows, NUM_ROWS * sizeof(unsigned int), hipMemcpyHostToDevice );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(dev_nim, dev_rows, dev_move);

        // Copy the move back from the GPU to the CPU
        hipMemcpy( move, dev_move, sizeof(Nimply), hipMemcpyDeviceToHost );

        // Free the memory allocated on the GPU
        hipFree( dev_rows );
        hipFree( dev_nim );
        hipFree( dev_move );

        nimming(nim, move);

        // Free the memory we allocated on the CPU
        free(move);

        printf("GPU Minmax: ");
        printRows(nim);

        // The CPU perform a random move
        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}

int main_CPU(void) {
    // remember to include ONLY the CPU library [nimlib/nimlib.h]

    Nim* nim = createNim(NUM_ROWS);
    printRows(nim);

    
    Nimply* move;
    while(isNotEnded(nim)) {
        move = minmax(nim);
        nimming(nim, move);
        destroyNimply(move);
        printf("Minmax: ");
        printRows(nim);

        if (isNotEnded(nim)) {
            randomStrategy(nim);
            printf("Random: ");
            printRows(nim);
        }
    }
    
    destroyNim(nim);
    return 0;
}