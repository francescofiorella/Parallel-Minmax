#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nim.h"
#include "./nimlib/agents.h"

#define NUM_ROWS 3

int main(void) {
    Nim* nim = createNim(NUM_ROWS);
    printRows(nim);

    randomStrategy(nim);

    printRows(nim);

    destroyNim(nim);
    return 0;
}
