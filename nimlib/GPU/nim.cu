#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nim.cuh"

__host__ __device__ void printNimply(Nimply* nimply) {
    printf("Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

void destroyNim(Nim* nim) {
    if (nim) {
        if (nim->rows) {
            free(nim->rows);
        }
        free(nim);
    }
}

void createNim(Nim* output, unsigned int* rows, unsigned int numRows) {
    // unsigned int rows[numRows]; // check if it is ok or it needs to be passed as argument
    output->numRows = numRows;
    output->turn = 0;
    for (int i = 0; i < numRows; i++) {
        rows[i] = i * 2 + 1;
    }
    output->rows = rows;
}

__device__ void deepcopyNim(Nim* nim, Nim* output, unsigned int* outputRows) {
    output->numRows = nim->numRows;
    output->turn = nim->turn;
    for (int i = 0; i < nim->numRows; i++) {
        outputRows[i] = nim->rows[i];
    }
    output->rows = outputRows;
}

__host__ __device__ bool isNotEnded(Nim* nim) {
    unsigned int sum = 0;
    for (int i = 0; i < nim->numRows; i++) {
        sum = sum + nim->rows[i];
    }
    return sum != 0;
}

__host__ __device__ void printRows(Nim* nim) {
    printf("Rows: %d", nim->rows[0]);
    for (int i = 1; i < nim->numRows; i++) {
        printf(", %d", nim->rows[i]);
    }
    printf("\n");
}

__host__ __device__ void nimming(Nim* nim, Nimply* nimply) {
    if (nim->numRows <= nimply->row) {
        printf("Not enougth rows!\n");
        return;
    }
    if (nim->rows[nimply->row] < nimply->numSticks) {
        printf("Not enougth sticks!\n");
        return;
    }
    if (nimply->numSticks < 1) {
        printf("Not a valid move!\n");
        return;
    }
    nim->rows[nimply->row] = nim->rows[nimply->row] - nimply->numSticks;
    nim->turn = 1 - nim->turn;
}

__host__ __device__ void possibleMoves(Nim* nim, MovesArray* output) {
    unsigned int index = 0;
    for (int r = 0; r < nim->numRows; r++) {
        unsigned int c = nim->rows[r];
        for (int o = 1; o <= c; o++) {
            Nimply ply;
            ply.row = r;
            ply.numSticks = o;
            output->array[index] = ply;
            index++;
        }
    }
    output->numItems = index;
}
