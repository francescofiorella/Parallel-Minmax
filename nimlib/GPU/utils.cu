#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__device__ void printResult(Result* result) {
    if (!result) {
        printf("Result: NULL");
        return;
    }
    printf("Result:\n");
    printNimply(&(result->ply));
    printf("Val: %d\n", result->val);
}

__device__ void printResultArray(ResultArray* resultArray) {}
__device__ void printEntry(StackEntry* entry) {}

__device__ void resultArrayPush(ResultArray* resultArray, unsigned int maxSize, Nimply* ply, int val) {
    if (resultArray->numItems == maxSize) {
        printf("the resultArray size is not enougth!\n");
        return;
    }
    unsigned int index = resultArray->numItems;
    resultArray->array[index].ply = *ply;
    resultArray->array[index].val = val;
    resultArray->numItems++;
}

__device__ void minResultArray(ResultArray* results, Result* output) {
    if (results->numItems < 1) {
        printf("empty resultArray!\n");
        return;
    }
    unsigned int min_index = 0;
    int min_val = results->array[0].val;
    for (int i = 1; i < results->numItems; i++) {
        if ((results->array[i]).val < min_val) {
            min_index = i;
            min_val = (results->array[i]).val;
        }
    }
    output->ply = (results->array[min_index]).ply;
    output->val = min_val;
}

__device__ void maxResultArray(ResultArray* results, Result* output) {
    if (results->numItems < 1) {
        printf("empty ResultArray!\n");
        return;
    }
    unsigned int max_index = 0;
    int max_val = results->array[0].val;
    for (int i = 1; i < results->numItems; i++) {
        if ((results->array[i]).val > max_val) {
            max_index = i;
            max_val = (results->array[i]).val;
        }
    }
    output->ply = (results->array[max_index]).ply;
    output->val = max_val;
}

__device__ void stackPush(Stack* stack, unsigned int maxStackSize, Nim* board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, ResultArray* evaluations, Result* result) {
    if (stack->stackSize == maxStackSize) {
        printf("the stack size is not enougth!\n");
        return;
    }
    unsigned int index = stack->stackSize;
    stack->array[index].board = board;
    stack->array[index].alpha = alpha;
    stack->array[index].beta = beta;
    stack->array[index].player = player;
    stack->array[index].depth = depth;
    stack->array[index].plyIndex = plyIndex;
    stack->array[index].stackIndex = stackIndex;
    stack->array[index].evaluations = evaluations;
    stack->array[index].result = result;
    stack->stackSize++;
}

__device__ void stackPop(Stack* stack, StackEntry* entry) {
    if (stack->stackSize < 1) {
        printf("the stack is empty!\n");
        return;
    }
    stack->stackSize--;
    unsigned int index = stack->stackSize;
    entry->board = stack->array[index].board;
    entry->alpha = stack->array[index].alpha;
    entry->beta = stack->array[index].beta;
    entry->player = stack->array[index].player;
    entry->depth = stack->array[index].depth;
    entry->plyIndex = stack->array[index].plyIndex;
    entry->stackIndex = stack->array[index].stackIndex;
    entry->evaluations = stack->array[index].evaluations;
    entry->result = stack->array[index].result;
    printf("Inner evaluations:\n");
    printf("%d\n", entry->evaluations->numItems);
}
