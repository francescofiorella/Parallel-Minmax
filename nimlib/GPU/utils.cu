#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__device__ void resultArrayPush(ResultArray* resultArray, unsigned int maxSize, Result* result) {
    if (resultArray->numItems == maxSize) {
        printf("the resultArray size is not enougth!\n");
        return;
    }
    resultArray->array[resultArray->numItems] = *result;
    resultArray->numItems++;
}

__device__ void minResultArray(ResultArray* results, Result* output) {
    if (results->numItems < 1) {
        printf("empty resultArray!\n");
        return;
    }
    output = &(results->array[0]);
    for (int i = 1; i < results->numItems; i++) {
        if ((results->array[i]).val < output->val) {
            output = &(results->array[i]);
        }
    }
}

__device__ void maxResultArray(ResultArray* resultArray, Result* output) {
    if (resultArray->numItems < 1) {
        printf("empty ResultArray!\n");
        return;
    }
    output = &(resultArray->array[0]);
    for (int i = 1; i < resultArray->numItems; i++) {
        if ((resultArray->array[i]).val > output->val) {
            output = &(resultArray->array[i]);
        }
    }
}

__device__ void createStackEntry(StackEntry* entry, Nim* board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, ResultArray* evaluations, Result* result) {
    entry->board = board;
    entry->alpha = alpha;
    entry->beta = beta;
    entry->player = player;
    entry->depth = depth;
    entry->plyIndex = plyIndex;
    entry->stackIndex = stackIndex;
    entry->evaluations = evaluations;
    entry->result = result;
}

__device__ void stackPush(Stack* stack, unsigned int maxStackSize, StackEntry* stackEntry) {
    if (stack->stackSize == maxStackSize) {
        printf("the stack size is not enougth!\n");
        return;
    }
    stack->array[stack->stackSize] = *stackEntry;
    stack->stackSize++;
}

__device__ void stackPop(Stack* stack, StackEntry* entry) {
    if (stack->stackSize < 1) {
        printf("the stack is empty!\n");
        return;
    }
    stack->stackSize--;
    entry = &(stack->array[stack->stackSize]);
}
