#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nimlib.cuh"

__global__ void GPU_minmax(Nim* nim, unsigned int* rows, ResultArray* results, Result* resultArray, MovesArray* moves, Nimply* plys, Nimply* ply) {
    // Associate arrays to classes
    nim->rows = rows;
    results->array = resultArray;
    moves->array = plys;

    const unsigned int numRows = 5;
    const unsigned int maxMoves = 25; 

    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned int stopComputation = 0;
    // stopComputation values:
    // 0 - Keep calculating
    // 1 - Calculate only the global result
    // 2 - Calculate the shared and the global result
    
    // if (bid == 0 && tid == 0) {
    //     // initialize the global output
    //     // the max number of results is equal to the available moves => the max num of moves is rows^2
        
    //     // calculate the first moves
    //     possibleMoves(nim, moves);
    //     results->numItems = moves->numItems;
    // }
    
    // __syncthreads();
    
    if (bid >= moves->numItems)
        return;
    
    __syncthreads();
    
    __shared__ MovesArray sharedMoves;
    __shared__ Nimply sharedPlys[maxMoves];
    sharedMoves.array = sharedPlys;
    
    __shared__ Nim sharedBoard;
    __shared__ unsigned int sharedRows[numRows];
    __shared__ int sharedPlayer;
    sharedPlayer = 1;
    if (tid == 0) {
        // calculate the new board and invert the current player
        deepcopyNim(nim, &sharedBoard, sharedRows);
        // select the move from bid
        // calculate the resulting board for the current move
        nimming(&sharedBoard, &(moves->array[bid]));
        sharedPlayer = 1 - sharedPlayer;

        // check if the game is ended, if yes update the results
        if (!isNotEnded(&sharedBoard)) {
            Result res;
            res.ply = moves->array[bid];
            res.val = sharedPlayer;
            results->array[bid] = res;

            // jump to min/max ending evaluation if bid == 0 and tid == 0
            if (bid == 0)
                stopComputation = 1;
        }

        // calculate the new moves on shared array
        possibleMoves(&sharedBoard, &sharedMoves);
    }

    __syncthreads();

    // works also if nim is ended
    if (stopComputation == 0 && tid >= sharedMoves.numItems)
        return;

    // __syncthreads();

    Nim newBoard;
    int player = sharedPlayer;
    __shared__ ResultArray sharedResults;
    __shared__ Result sharedResultArray[maxMoves];
    if (stopComputation == 0) {
        sharedResults.array = sharedResultArray;
        sharedResults.numItems = sharedMoves.numItems;

        // declare Nim for this thread
        unsigned int newRows[numRows];
        deepcopyNim(&sharedBoard, &newBoard, newRows);
        // apply tid move
        nimming(&newBoard, &(sharedMoves.array[tid]));
        player = 1 - player;

        // check if nim is ended
        if (!isNotEnded(&newBoard)) {
            Result res;
            res.ply = sharedMoves.array[tid];
            res.val = player;
            sharedResults.array[tid] = res;
            
            if (tid != 0)
                return;

            // stop the kernel only if tid != 0 else evaluate all the shared
            stopComputation = 2;
        }
    }

    __syncthreads();

    if (stopComputation == 0) {
        // start to calculate the minmax, store the result in sharedResults
        if (bid == 0 && tid == 0) {
            standard_minmax(&newBoard, player, tid, sharedResults.array);
            printNimply(&(sharedResults.array[0].ply));
        }
        // standard_minmax(&newBoard, player, tid, sharedResults.array);

        if (tid != 0)
            return;
    }

    // when all secondary threads finished
    __syncthreads();

    if (stopComputation != 1) {
        // calculate the best move from the shared results
        Result sharedResult;
        maxResultArray(&sharedResults, &sharedResult);
        results->array[bid] = sharedResult;

        if (bid != 0)
            return;
    }

    __syncthreads();

    // calculate the best move from the global results
    Result lastResult;
    minResultArray(results, &lastResult);
    ply->row = lastResult.ply.row;
    ply->numSticks = lastResult.ply.numSticks;
}

// let's remove alpha beta pruning and max depth constrains
// push in the stack every move, using the same evaluations pointer
// at every move, push every move below
// use the depth value to discriminate between layers
// until 1024 (!?)

// modify the algorithm in order to perform one evaluation per thread, or
// use the same algorithm but run it several times in different threads



// bid e tid 0
// calcola mosse per nim originale
// inizializza vettore risultati [results]

// un bid per ogni mossa [25]

// tid 0
// applica mossa bid
// controlla se terminato -> se si, aggiungi a results
// inverti player
// calcola mosse per nuova board

// tid da 0 a 25
// applica mossa tid
// controlla se terminato -> se si, aggiungi a [?]
// inverti player
// fai partire loop per ogni thread

// sharedResults is the output
__device__ void standard_minmax(Nim* nim, int player, unsigned int tid, Result* sharedResults) {
    const unsigned int maxStackSize = 1000; /* TODO */
    const unsigned int numRows = 5;
    const unsigned int maxMoves = 25; 

    // init the stack
    Stack stack;
    stack.stackSize = 0;
    StackEntry entries[maxStackSize];
    stack.array = entries;
    
    // push the very first empty entry
    StackEntry entry;
    createStackEntry(&entry, NULL, 0, 0, 0, 0, 0, 0, NULL, NULL);
    stackPush(&stack, maxStackSize, &entry);

    // push the first meaningful entry
    Nim newBoard;
    unsigned int newRows[numRows];
    deepcopyNim(nim, &newBoard, newRows);
    ResultArray evaluations;
    Result evaluationsArray[maxMoves];
    evaluations.array = evaluationsArray;
    evaluations.numItems = 0;
    createStackEntry(&entry, &newBoard, -1, 1, 1, 0, -1, stack.stackSize-1, &evaluations, NULL);
    stackPush(&stack, maxStackSize, &entry);

    // while there are moves to evaluate
    while (stack.stackSize > 1) {
        stackPop(&stack, &entry);

        // stop if the game ended
        if (!isNotEnded(entry.board)) {
            Result res;
            res.val = entry.player;
            stack.array[entry.stackIndex].result = &res;
            continue;
        }
        // calculate the posible moves
        MovesArray moves;
        Nimply plys[maxMoves];
        moves.array = plys;
        possibleMoves(entry.board, &moves);
        // use the calculated result if it's not the first move
        if (entry.plyIndex != -1) {
            Result result;
            result.ply = moves.array[entry.plyIndex];
            // exploit the previous result calculation
            int val = entry.result->val;
            result.val = val;
            resultArrayPush(entry.evaluations, maxMoves, &result);
            // update alpha or beta
            if (entry.player == 1) {
                if (entry.beta > val) entry.beta = val;
            } else {
                if (entry.alpha < val) entry.alpha = val;
            }
            // stop if it's the last move or it's time to prune
            if (entry.plyIndex == moves.numItems - 1 || entry.beta <= entry.alpha) {
                Result r;
                if (entry.player == 1) {
                    minResultArray(entry.evaluations, &r);
                } else {
                    maxResultArray(entry.evaluations, &r);
                }
                stack.array[entry.stackIndex].result = &r;
                continue;
            }
        }
        // evaluate the next move
        unsigned int newRows_[numRows];
        deepcopyNim(entry.board, &newBoard, newRows_);
        nimming(&newBoard, &(moves.array[entry.plyIndex+1]));
        // push the previous state
        StackEntry newEntry;
        createStackEntry(&newEntry, entry.board, entry.alpha, entry.beta, entry.player, entry.depth, entry.plyIndex + 1, entry.stackIndex, entry.evaluations, entry.result);
        stackPush(&stack, maxStackSize, &newEntry);
        // push the current state (after making the move)
        ResultArray evaluations_;
        Result evaluationsArray_[maxMoves];
        evaluations_.array = evaluationsArray_;
        evaluations_.numItems = 0;
        createStackEntry(&newEntry, &newBoard, entry.alpha, entry.beta, -(entry.player), entry.depth + 1, -1, stack.stackSize - 1, &evaluations_, NULL);
        stackPush(&stack, maxStackSize, &newEntry);
    }
    stackPop(&stack, &entry);
    // push the result into the shared results
    sharedResults[tid] = *(entry.result);
}

void randomStrategy(Nim* nim) {
    MovesArray* moves;
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    if (!moves) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    unsigned int maxMoves = nim->numRows * nim->numRows;
    Nimply array[maxMoves];
    moves->array = array;
    possibleMoves(nim, moves);

    if (moves->numItems < 1) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % moves->numItems;
    nimming(nim, &(moves->array[r]));

    free(moves);
}
