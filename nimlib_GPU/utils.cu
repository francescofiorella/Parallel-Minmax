#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__device__ void resultArrayPush(ResultArray* resultArray, unsigned int maxSize, Result* result) {
    if (resultArray->numItems == maxSize) {
        fprintf(stderr, "the resultArray size is not enougth!\n");
        exit(1);
    }
    resultArray->array[resultArray->numItems] = result;
    resultArray->numItems++;
}

__device__ void minResultArray(ResultArray* results, Result* output) {
    if (resultArray->numItems < 1) {
        fprintf(stderr, "empty ResultArray!\n");
        exit(1);
    }
    output = resultArray->array[0];
    for (int i = 1; i < resultArray->numItems; i++) {
        if ((resultArray->array[i])->val < output->val) {
            output = (resultArray->array[i]);
        }
    }
}

__device__ void maxResultArray(ResultArray* resultArray, Result* output) {
    if (resultArray->numItems < 1) {
        fprintf(stderr, "empty ResultArray!\n");
        exit(1);
    }
    output = resultArray->array[0];
    for (int i = 1; i < resultArray->numItems; i++) {
        if ((resultArray->array[i])->val > output->val) {
            output = (resultArray->array[i]);
        }
    }
}

__device__ void createStackEntry(StackEntry* entry, Nim* board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, ResultArray* evaluations, Result* result) {
    entry->board = board;
    entry->alpha = alpha;
    entry->beta = beta;
    entry->player = player;
    entry->depth = depth;
    entry->plyIndex = plyIndex;
    entry->stackIndex = stackIndex;
    entry->evaluations = evaluations;
    entry->result = result;
}

__device__ void stackPush(Stack* stack, unsigned int maxStackSize, StackEntry* stackEntry) {
    if (stack->stackSize == maxStackSize) {
        fprintf(stderr, "the stack size is not enougth!\n");
        exit(1);
    }
    stack->array[stack->stackSize] = stackEntry;
    stack->stackSize++;
}

__device__ void stackPop(Stack* stack, StackEntry* entry) {
    if (stack->stackSize < 1) {
        fprintf(stderr, "the stack is empty!\n");
        exit(1);
    }
    stack->stackSize--;
    entry = stack->array[stack->stackSize];
}
