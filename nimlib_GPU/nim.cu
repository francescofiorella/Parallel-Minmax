#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "nim.cuh"

void printNimply(Nimply* nimply) {
    printf("Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

void destroyNim(Nim* nim) {
    if (nim) {
        if (nim->rows) {
            free(nim->rows);
        }
        free(nim);
    }
}

void createNim(Nim* output, unsigned int numRows) {
    unsigned int rows[numRows]; // check if it is ok or it needs to be passed as argument
    output->numRows = numRows;
    output->turn = 0;
    output->rows = rows;
    for (int i = 0; i < numRows; i++) {
        output->rows[i] = i * 2 + 1;
    }
}

__device__ void deepcopyNim(Nim* nim, Nim* output, unsigned int* outputRows) {
    output->numRows = nim->numRows;
    output->turn = nim->turn;
    output->rows = outputRows;
    for (int i = 0; i < nim->numRows; i++) {
        output->rows[i] = nim->rows[i];
    }
}

__device__ bool isNotEnded(Nim* nim) {
    unsigned int sum = 0;
    for (int i = 0; i < nim->numRows; i++) {
        sum = sum + nim->rows[i];
    }
    return sum != 0;
}

bool isNotEnded(Nim* nim) {
    unsigned int sum = 0;
    for (int i = 0; i < nim->numRows; i++) {
        sum = sum + nim->rows[i];
    }
    return sum != 0;
}

void printRows(Nim* nim) {
    printf("Rows: %d", nim->rows[0]);
    for (int i = 1; i < nim->numRows; i++) {
        printf(", %d", nim->rows[i]);
    }
    printf("\n");
}

__device__ void nimming(Nim* nim, Nimply* nimply) {
    if (nim->numRows <= nimply->row) {
        fprintf(stderr, "Not enougth rows!\n");
        exit(1);
    }
    if (nim->rows[nimply->row] < nimply->numSticks) {
        fprintf(stderr, "Not enougth sticks!\n");
        exit(1);
    }
    if (nimply->numSticks < 1) {
        fprintf(stderr, "Not a valid move!\n");
        exit(1);
    }
    nim->rows[nimply->row] = nim->rows[nimply->row] - nimply->numSticks;
    nim->turn = 1 - nim->turn;
}

__device__ void possibleMoves(Nim* nim, MovesArray* output) {
    unsigned int index = 0;
    for (int r = 0; r < nim->numRows; r++) {
        unsigned int c = nim->rows[r];
        for (int o = 1; o <= c; o++) {
            Nimply ply;
            ply.row = r;
            ply.numSticks = o;
            output->array[index] = ply;
            index++;
        }
    }
    output->numItems = index;
}
