#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nim.cuh"

__host__ __device__ void printNimply(Nimply* nimply) {
    printf("Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

void destroyNim(Nim* nim) {
    if (nim) {
        if (nim->rows) {
            free(nim->rows);
        }
        free(nim);
    }
}

void createNim(Nim* output, unsigned int* rows, unsigned int numRows) {
    // unsigned int rows[numRows]; // check if it is ok or it needs to be passed as argument
    output->numRows = numRows;
    output->turn = 0;
    output->rows = rows;
    for (int i = 0; i < numRows; i++) {
        output->rows[i] = i * 2 + 1;
    }
}

__device__ void deepcopyNim(Nim* nim, Nim* output, unsigned int* outputRows) {
    output->numRows = nim->numRows;
    output->turn = nim->turn;
    output->rows = outputRows;
    for (int i = 0; i < nim->numRows; i++) {
        output->rows[i] = nim->rows[i];
    }
}

__host__ __device__ bool isNotEnded(Nim* nim) {
    unsigned int sum = 0;
    for (int i = 0; i < nim->numRows; i++) {
        sum = sum + nim->rows[i];
    }
    return sum != 0;
}

__host__ __device__ void printRows(Nim* nim) {
    printf("Rows: %d", nim->rows[0]);
    for (int i = 1; i < nim->numRows; i++) {
        printf(", %d", nim->rows[i]);
    }
    printf("\n");
}

__host__ __device__ void nimming(Nim* nim, Nimply* nimply) {
    if (nim->numRows <= nimply->row) {
        printf("Not enougth rows!\n");
        return;
    }
    if (nim->rows[nimply->row] < nimply->numSticks) {
        printf("Not enougth sticks!\n");
        return;
    }
    if (nimply->numSticks < 1) {
        printf("Not a valid move!\n");
        return;
    }
    nim->rows[nimply->row] = nim->rows[nimply->row] - nimply->numSticks;
    nim->turn = 1 - nim->turn;
}

__host__ __device__ void possibleMoves(Nim* nim, MovesArray* output) {
    unsigned int index = 0;
    for (int r = 0; r < nim->numRows; r++) {
        unsigned int c = nim->rows[r];
        for (int o = 1; o <= c; o++) {
            Nimply ply;
            ply.row = r;
            ply.numSticks = o;
            output->array[index] = ply;
            index++;
        }
    }
    output->numItems = index;
}

void randomStrategy(Nim* nim) {
    MovesArray* moves;
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    if (!moves) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    unsigned int maxMoves = nim->numRows * nim->numRows;
    Nimply array[maxMoves];
    moves->array = array;
    possibleMoves(nim, moves);

    if (moves->numItems < 1) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % moves->numItems;
    nimming(nim, &(moves->array[r]));

    free(moves);
}
