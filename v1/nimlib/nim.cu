#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nim.cuh"

__host__ __device__ void printNimply(Nimply* nimply) {
    if (!nimply) {
        printf("Nimply - NULL\n");
        return;
    }
    printf("Nimply - Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

__host__ __device__ void printNim(Nim* nim) {
    if (!nim || ! nim->rows) {
        printf("Nim - NULL");
        return;
    }
    printf("Nim - %d", nim->rows[0]);
    for (int i = 1; i < nim->numRows; i++) {
        printf(", %d", nim->rows[i]);
    }
    printf("\n");
}

__device__ void printMovesArray(MovesArray* movesArray) {
    if (!movesArray || !movesArray->array) {
        printf("MovesArray - NULL\n");
        return;
    }
    if (movesArray->numItems == 0) {
        printf("MovesArray - void");
        return;
    }
    printf("MovesArray - [\n");
    for (int i = 0; i < movesArray->numItems; i++) {
        printf("   Row: %d, Num: %d\n", movesArray->array[i].row, movesArray->array[i].numSticks);
    }
    printf("]\n");
}

void createNim(Nim* output, unsigned int numRows) {
    output->numRows = numRows;
    for (int i = 0; i < numRows; i++) {
        output->rows[i] = i * 2 + 1;
    }
}

__device__ void deepcopyNim(Nim* nim, Nim* output) {
    output->numRows = nim->numRows;
    for (int i = 0; i < nim->numRows; i++) {
        output->rows[i] = nim->rows[i];
    }
}

__host__ __device__ bool isNotEnded(Nim* nim) {
    unsigned int sum = 0;
    for (int i = 0; i < nim->numRows; i++) {
        sum = sum + nim->rows[i];
    }
    return sum != 0;
}

__host__ __device__ void nimming(Nim* nim, Nimply* nimply) {
    if (nim->numRows <= nimply->row) {
        printf("Not enough rows!\n");
        return;
    }
    if (nim->rows[nimply->row] < nimply->numSticks) {
        printf("Not enough sticks!\n");
        return;
    }
    if (nimply->numSticks < 1) {
        printf("Not a valid move!\n");
        return;
    }
    nim->rows[nimply->row] = nim->rows[nimply->row] - nimply->numSticks;
}

__host__ __device__ void possibleMoves(Nim* nim, MovesArray* output) {
    unsigned int index = 0;
    for (int r = 0; r < nim->numRows; r++) {
        unsigned int c = nim->rows[r];
        for (int o = 1; o <= c; o++) {
            Nimply ply;
            ply.row = r;
            ply.numSticks = o;
            output->array[index] = ply;
            index++;
        }
    }
    output->numItems = index;
}

__host__ __device__ bool nim_sum(Nim* nim) {
    int nim_sum = nim->rows[0];
    for (int i = 1; i < nim->numRows; i++) {
        nim_sum ^= nim->rows[i];
    }
    return nim_sum == 0; 
}
