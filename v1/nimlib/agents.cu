#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nimlib.cuh"

#define NUM_MOVES 25 // NUM_ROWS*NUM_ROW

__global__ void GPU_minmax(Nim* nim, MovesArray* moves, Nimply* plys, ResultArray* results, Result* resultArray) {
    // Associate arrays to classes
    results->array = resultArray;
    results->numItems = moves->numItems;
    moves->array = plys;

    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned int stopComputation = 0;
    // stopComputation values:
    // 0 - Keep calculating
    // 1 - Calculate only the global result
    // 2 - Calculate the shared and the global result
    
    if (bid >= moves->numItems)
        return;
    
    __syncthreads();
    
    __shared__ MovesArray sharedMoves;
    __shared__ Nimply sharedPlys[NUM_MOVES];
    sharedMoves.array = sharedPlys;
    
    __shared__ Nim sharedBoard;
    __shared__ int sharedPlayer;
    sharedPlayer = 1;
    if (tid == 0) {
        // calculate the new board and invert the current player
        deepcopyNim(nim, &sharedBoard);
        // select the move from bid
        // calculate the resulting board for the current move
        nimming(&sharedBoard, &(moves->array[bid]));
        sharedPlayer = -sharedPlayer;

        // check if the game is ended, if yes update the results
        if (!isNotEnded(&sharedBoard)) {
            Result res;
            res.ply = moves->array[bid];
            res.val = sharedPlayer;
            results->array[bid] = res;

            // jump to min/max ending evaluation if bid == 0 and tid == 0
            if (bid == 0)
                stopComputation = 1;
        }

        // calculate the new moves on shared array
        possibleMoves(&sharedBoard, &sharedMoves);
    }

    __syncthreads();

    // works also if nim is ended
    if (stopComputation == 0 && tid >= sharedMoves.numItems)
        return;

    Nim newBoard;
    int player = sharedPlayer;
    __shared__ ResultArray sharedResults;
    __shared__ Result sharedResultArray[NUM_MOVES];
    if (stopComputation == 0) {
        sharedResults.array = sharedResultArray;
        sharedResults.numItems = sharedMoves.numItems;

        // declare Nim for this thread
        deepcopyNim(&sharedBoard, &newBoard);
        // apply tid move
        nimming(&newBoard, &(sharedMoves.array[tid]));
        player = -player;

        // check if nim is ended
        if (!isNotEnded(&newBoard)) {
            Result res;
            res.ply = sharedMoves.array[tid];
            res.val = player;
            sharedResults.array[tid] = res;
            
            if (tid != 0)
                return;

            // stop the kernel only if tid != 0 else evaluate all the shared
            stopComputation = 2;
        }
    }

    __syncthreads();

    if (stopComputation == 0) {
        // start to calculate the minmax, store the result in sharedResults
        standard_minmax(&newBoard, player, tid, sharedResults.array);

        if (tid != 0)
            return;
    }

    // when all secondary threads finished
    __syncthreads();

    if (stopComputation != 1) {
        // calculate the best move from the shared results
        Result sharedResult;
        maxResultArray(&sharedResults, &sharedResult);
        results->array[bid] = sharedResult;

        if (bid != 0)
            return;
    }
}

// sharedResults is the output
__device__ void standard_minmax(Nim* nim, int player, unsigned int tid, Result* sharedResults) {
    const unsigned int maxDepth = 5;
    const unsigned int maxStackSize = 8;
    /*
    | Max Depth | Max Stack Size |
    | --------- | -------------- |
    | 1         | 4              |
    | 2         | 5              |
    | 3         | 6              |
    | 4         | 7              |
    | 5         | 8              |
    | 6         | 9              |
    | 7         | 10             |
    */

    // init the stack
    Stack stack;
    stack.stackSize = 0;
    StackEntry entries[maxStackSize];
    stack.array = entries;
    
    // push the very first empty entry
    stackPush(&stack, maxStackSize, NULL, 0, 0, 0, 0, 0, 0, NULL, NULL);

    // push the first meaningful entry
    Nim newBoard;
    deepcopyNim(nim, &newBoard);
    ResultArray evaluations;
    Result evaluationsArray[NUM_MOVES];
    evaluations.array = evaluationsArray;
    evaluations.numItems = 0;
    stackPush(&stack, maxStackSize, &newBoard, -1, 1, 1, 0, -1, stack.stackSize-1, &evaluations, NULL);

    StackEntry entry;
    
    // while there are moves to evaluate
    while (stack.stackSize > 1) {
        __syncthreads();

        stackPop(&stack, &entry);

        // stop if the max depth was reached
        if (entry.depth > maxDepth) {
            Result res;
            // res.val = nim_sum(&(entry.board)) ? entry.player : -entry.player;
            res.val = -entry.player;
            stack.array[entry.stackIndex].result = res;
            continue;
        }
        // stop if the game ended
        if (!isNotEnded(&(entry.board))) {
            Result res;
            res.val = entry.player;
            stack.array[entry.stackIndex].result = res;
            continue;
        }
        // calculate the posible moves
        MovesArray moves;
        Nimply plys[NUM_MOVES];
        moves.array = plys;
        possibleMoves(&(entry.board), &moves);
        __syncthreads();
        // use the calculated result if it's not the first move
        if (entry.plyIndex != -1) {
            // exploit the previous result calculation
            int val = entry.result.val;
            resultArrayPush(&(entry.evaluations), NUM_MOVES, &(moves.array[entry.plyIndex]), val);
            // update alpha or beta
            if (entry.player == 1) {
                if (entry.beta > val) entry.beta = val;
            } else {
                if (entry.alpha < val) entry.alpha = val;
            }
            // stop if it's the last move or it's time to prune
            if (entry.plyIndex == moves.numItems - 1 || entry.beta <= entry.alpha) {
                Result r;
                if (entry.player == 1) {
                    minResultArray(&(entry.evaluations), &r);
                } else {
                    maxResultArray(&(entry.evaluations), &r);
                }
                stack.array[entry.stackIndex].result = r;
                continue;
            }
        }
        __syncthreads();
        // evaluate the next move
        deepcopyNim(&(entry.board), &newBoard);
        nimming(&newBoard, &(moves.array[entry.plyIndex+1]));
        // push the previous state
        stackPush(&stack, maxStackSize, &(entry.board), entry.alpha, entry.beta, entry.player, entry.depth, entry.plyIndex + 1, entry.stackIndex, &(entry.evaluations), &(entry.result));
        __syncthreads();
        // push the current state (after making the move)
        ResultArray evaluations_;
        Result evaluationsArray_[NUM_MOVES];
        evaluations_.array = evaluationsArray_;
        evaluations_.numItems = 0;
        stackPush(&stack, maxStackSize, &newBoard, entry.alpha, entry.beta, -(entry.player), entry.depth + 1, -1, stack.stackSize - 1, &evaluations_, NULL);
    }
    stackPop(&stack, &entry);
    // printEntry(&entry);
    // push the result into the shared results
    sharedResults[tid] = entry.result;
}

void randomStrategy(Nim* nim, bool print) {
    MovesArray* moves;
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    if (!moves) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    unsigned int numMoves = nim->numRows * nim->numRows;
    Nimply array[numMoves];
    moves->array = array;
    possibleMoves(nim, moves);

    if (moves->numItems < 1) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % moves->numItems;
    Nimply* ply = &(moves->array[r]);
    nimming(nim, ply);
    if (print){
        printf("Random - (%d, %d)\n", ply->row, ply->numSticks);
        printNim(nim);
        printf("\n");
    } 

    free(moves);
}
