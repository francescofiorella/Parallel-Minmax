#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nimlib.cuh"

#define NUM_ROWS 5

#define cudaHandleError(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    Nim nim; // nim on the host CPU machine
    Nim* dev_nim; // nim for the GPU device

    MovesArray* moves; // the possible moves on the host CPU machine
    MovesArray* dev_moves; // the possible moves for the GPU device
    Nimply* dev_plys;
    ResultArray* results;
    ResultArray* dev_results;
    Result* dev_resultArray;

    // Initialize nim
    createNim(&nim, NUM_ROWS);
    printf("\n");
    printf("Initial board:\n");
    printNim(&nim);
    printf("\n");

    unsigned int player = 1;

    unsigned int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(&nim) && a == 0) {
        a++;
        // Allocate the memory on the CPU
        moves = (MovesArray*)malloc(sizeof(MovesArray));
        moves->numItems = 0;
        moves->array = (Nimply*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Nimply));
        results = (ResultArray*)malloc(sizeof(ResultArray));
        results->array = (Result*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Result));
        if (!results || !results->array || !moves || !moves->array) {
            fprintf(stderr, "malloc failure\n");
            exit(1);
        }

        // calculate the first level of the tree
        possibleMoves(&nim, moves);
        results->numItems = moves->numItems;

        // Allocate the memory on the GPU
        cudaHandleError( hipMalloc( (void**)&dev_nim, sizeof(Nim) ) );
        cudaHandleError( hipMalloc( (void**)&dev_moves, sizeof(MovesArray) ) );
        cudaHandleError( hipMalloc( (void**)&dev_plys, NUM_ROWS*NUM_ROWS * sizeof(Nimply) ) );
        cudaHandleError( hipMalloc( (void**)&dev_results, sizeof(ResultArray) ) );
        cudaHandleError( hipMalloc( (void**)&dev_resultArray, NUM_ROWS*NUM_ROWS * sizeof(Result) ) );

        // Copy nim to the GPU
        cudaHandleError( hipMemcpy( dev_nim, &nim, sizeof(Nim), hipMemcpyHostToDevice ) );

        cudaHandleError( hipMemcpy( dev_moves, moves, sizeof(MovesArray), hipMemcpyHostToDevice ) );
        cudaHandleError( hipMemcpy( dev_plys, moves->array, NUM_ROWS*NUM_ROWS * sizeof(Nimply), hipMemcpyHostToDevice ) );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(dev_nim, dev_moves, dev_plys, dev_results, dev_resultArray);
        
        cudaHandleError( hipPeekAtLastError() );

        // Copy the move back from the GPU to the CPU
        cudaHandleError( hipMemcpy( results->array, dev_resultArray, NUM_ROWS*NUM_ROWS * sizeof(Result), hipMemcpyDeviceToHost ) );

        // Free the memory allocated on the GPU
        hipFree( dev_nim );
        hipFree( dev_moves );
        hipFree( dev_plys );
        hipFree( dev_results );
        hipFree( dev_resultArray );
        
        // calculate the best move
        Nimply move;
        Result lastResult;
        minResultArray(results, &lastResult);
        move.row = lastResult.ply.row;
        move.numSticks = lastResult.ply.numSticks;

        // Perform the move
        nimming(&nim, &move);
        player = 1 - player;

        printf("GPU Minmax - (%d, %d)\n", move.row, move.numSticks);
        printNim(&nim);
        printf("\n");

        // Free the memory we allocated on the CPU
        free(moves->array);
        free(moves);
        free(results->array);
        free(results);

        // The CPU perform a random move
        if (isNotEnded(&nim)){
            randomStrategy(&nim, true);
            player = 1 - player;
        }
    }

    printf(player == 0 ? "Minmax won!\n" : "Random won!\n");

    return 0;
}
