#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <cstdio>
#include "nim.cuh"

__device__ Nimply* GPU_createNimply(unsigned int row, unsigned int numSticks) {
    Nimply* nimply;
    nimply = (Nimply*)malloc(sizeof(Nimply));
    nimply->row = row;
    nimply->numSticks = numSticks;
    return nimply;
}

__device__ void GPU_destroyNimply(Nimply* nimply) {
    if (nimply) {
        free(nimply);
    }
}

__device__ void GPU_printNimply(Nimply* nimply) {
    printf("Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

__device__ Nim* GPU_deepcopyNim(Nim* nim) {
    Nim* copy;
    copy = (Nim*)malloc(sizeof(Nim));
    copy->numRows = nim->numRows;
    copy->turn = nim->turn;
    copy->rows = (unsigned int*)malloc(nim->numRows * sizeof(unsigned int));
    if (!copy->rows) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    for (int i = 0; i < nim->numRows; i++) {
        copy->rows[i] = nim->rows[i];
    }
    return copy;
}

__device__ void GPU_printRows(Nim* nim) {
    printf("Rows: %d", nim->rows[0]);
    for (int i = 1; i < nim->numRows; i++) {
        printf(", %d", nim->rows[i]);
    }
    printf("\n");
}

__device__ void GPU_nimming(Nim* nim, Nimply* nimply) {
    if (nim->numRows <= nimply->row) {
        fprintf(stderr, "Not enougth rows!\n");
        exit(1);
    }
    if (nim->rows[nimply->row] < nimply->numSticks) {
        fprintf(stderr, "Not enougth sticks!\n");
        exit(1);
    }
    if (nimply->numSticks < 1) {
        fprintf(stderr, "Not a valid move!\n");
        exit(1);
    }
    nim->rows[nimply->row] = nim->rows[nimply->row] - nimply->numSticks;
    nim->turn = 1 - nim->turn;
}

__device__ MovesArray* GPU_possibleMoves(Nim* nim) {
    MovesArray* moves;
    moves = (MovesArray*)malloc(sizeof(MovesArray));

    unsigned int maxMoves = nim->numRows * nim->numRows;
    moves->array = (Nimply**)malloc(maxMoves * sizeof(Nimply*));
    if (!moves->array) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    unsigned int index = 0;
    for (int r = 0; r < nim->numRows; r++) {
        unsigned int c = nim->rows[r];
        for (int o = 1; o <= c; o++) {
            moves->array[index] = GPU_createNimply(r, o);
            index++;
        }
    }
    moves->numItems = index;
    return moves;
}

__device__ void GPU_destroyMovesArray(MovesArray* moves) {
    if (moves) {
        for (int i = 0; i < moves->numItems; i++) {
            GPU_destroyNimply(moves->array[i]);
        }
        if (moves->array) {
            free(moves->array);
        }
        free(moves);
    }
}

__device__ Result* GPU_createResult(Nimply* ply, int val) {
    Result* result;
    result = (Result*)malloc(sizeof(Result));
    if (!result) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    result->ply = ply;
    result->val = val;
    return result;
}

__device__ void GPU_destroyResult(Result* result) {
    if (result) {
        GPU_destroyNimply(result->ply);
        free(result);
    }
}

__device__ ResultArray* GPU_createResultArray(unsigned int maxSize) {
    ResultArray* resultArray;
    resultArray = (ResultArray*)malloc(sizeof(ResultArray));
    if (!resultArray) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    resultArray->maxSize = maxSize;
    resultArray->numItems = 0;
    resultArray->array = (Result**)malloc(maxSize * sizeof(Result*));
    if (!resultArray->array) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    return resultArray;
}

__device__ void GPU_destroyResultArray(ResultArray* resultArray) {
    if (resultArray) {
        if (resultArray->numItems > 0) {
            for (int i = resultArray->numItems + 1; i >= 0; i--) {
                GPU_destroyResult(resultArray->array[i]);
            }
        }
        if (resultArray->array) {
            free(resultArray->array);
        }
        free(resultArray);
    }
}

__device__ Result* GPU_minResultArray(ResultArray* resultArray) {
    if (resultArray->numItems < 1) {
        fprintf(stderr, "Empty ResultArray!\n");
        exit(1);
    }
    Result* min = resultArray->array[0];
    for (int i = 1; i < resultArray->numItems; i++) {
        if ((resultArray->array[i])->val < min->val) {
            min = (resultArray->array[i]);
        }
    }
    return min;
}

__device__ StackEntry* GPU_createStackEntry(Nim* board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, ResultArray* evaluations, Result* result) {
    StackEntry* stackEntry;
    stackEntry = (StackEntry*)malloc(sizeof(StackEntry));
    if (!stackEntry) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    stackEntry->board = board;
    stackEntry->alpha = alpha;
    stackEntry->beta = beta;
    stackEntry->player = player;
    stackEntry->depth = depth;
    stackEntry->plyIndex = plyIndex;
    stackEntry->stackIndex = stackIndex;
    stackEntry->evaluations = evaluations;
    stackEntry->result = result;
    return stackEntry;    
}

__device__ void GPU_destroyStackEntry(StackEntry* stackEntry) {
    if (stackEntry) {
        destroyNim(stackEntry->board);
        destroyResultArray(stackEntry->evaluations);
        destroyResult(stackEntry->result);
        free(stackEntry);
    }
}

__device__ Stack* GPU_createStack(unsigned int maxSize) {
    Stack* stack;
    stack = (Stack*)malloc(sizeof(Stack));
    if (!stack) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    stack->maxSize = maxSize;
    stack->stackSize = 0;
    stack->array = (StackEntry**)malloc(maxSize * sizeof(StackEntry*));
    if (!stack->array) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }
    return stack;
}

__device__ void GPU_destroyStack(Stack* stack) {
    if (stack) {
        if (stack->stackSize > 0) {
            for (int i = stack->stackSize - 1; i >= 0; i--) {
                destroyStackEntry(stack->array[i]);
            }
        }
        if (stack->array) {
            free(stack->array);
        }
        free(stack);
    }
}

__device__ void GPU_stackPush(Stack* stack, StackEntry* stackEntry) {
    /* TODO */
    // if (stack->stackSize == stack->maxSize) {
    //     // allocate more memory
    //     stack->maxSize = 2 * stack->maxSize;
    //     StackEntry** temp = stack->array;
    //     stack->array = (StackEntry**)malloc(stack->maxSize * sizeof(StackEntry*));
    //     if (!stack->array) {
    //         fprintf(stderr, "malloc failure\n");
    //         exit(1);
    //     }
    //     for (int i = stack->stackSize-1; i >= 0; i--) {
    //         stack->array[i] = temp[i];
    //     }
    //     free(temp);
    // }
    stack->array[stack->stackSize] = stackEntry;
    stack->stackSize++;
}

__device__ StackEntry* GPU_stackPop(Stack* stack) {
    if (stack-> stackSize < 1) {
        fprintf(stderr, "The stack is empty!\n");
        exit(1);
    }
    stack->stackSize--;
    StackEntry* entry = stack->array[stack->stackSize];
    stack->array[stack->stackSize] = NULL;
    return entry;
}
