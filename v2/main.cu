#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nimlib.cuh"

#define NUM_ROWS 5

#define cudaHandleError(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    Nimply* move; // the move on the host CPU machine
    Nimply* dev_move; // the move for the GPU device

    ResultArray* results;
    ResultArray* dev_results;
    Result* dev_resultArray;
    MovesArray* moves;
    MovesArray* dev_moves;
    Nimply* dev_plys;

    // Initialize nim
    unsigned int nim = createNim(NUM_ROWS);
    printf("\n");
    printf("Initial board:\n");
    printNim(nim, NUM_ROWS);
    printf("\n");

    unsigned int player = 1;

    unsigned int a = 0;
    // Execute the minmax on the GPU device iteratively, until the game ends
    while(isNotEnded(nim) && a == 0) {
        a++;
        // Allocate the memory on the CPU
        move = (Nimply*)malloc(sizeof(Nimply));
        results = (ResultArray*)malloc(sizeof(ResultArray));
        results->array = (Result*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Result));
        moves = (MovesArray*)malloc(sizeof(MovesArray));
        moves->numItems = 0;
        moves->array = (Nimply*)malloc(NUM_ROWS*NUM_ROWS * sizeof(Nimply));
        if (!move || !results || !results->array || !moves || !moves->array) {
            fprintf(stderr, "malloc failure\n");
            exit(1);
        }

        // calculate the first level of the tree
        possibleMoves(nim, NUM_ROWS, moves);
        results->numItems = moves->numItems;

        // Allocate the memory on the GPU
        cudaHandleError( hipMalloc( (void**)&dev_move, sizeof(Nimply) ) );
        cudaHandleError( hipMalloc( (void**)&dev_results, sizeof(ResultArray) ) );
        cudaHandleError( hipMalloc( (void**)&dev_resultArray, NUM_ROWS*NUM_ROWS * sizeof(Result) ) );
        cudaHandleError( hipMalloc( (void**)&dev_moves, sizeof(MovesArray) ) );
        cudaHandleError( hipMalloc( (void**)&dev_plys, NUM_ROWS*NUM_ROWS * sizeof(Nimply) ) );

        // Copy nim to the GPU
        cudaHandleError( hipMemcpy( dev_results, results, sizeof(ResultArray), hipMemcpyHostToDevice ) );
        cudaHandleError( hipMemcpy( dev_resultArray, results->array, NUM_ROWS*NUM_ROWS * sizeof(Result), hipMemcpyHostToDevice ) );
        cudaHandleError( hipMemcpy( dev_moves, moves, sizeof(MovesArray), hipMemcpyHostToDevice ) );
        cudaHandleError( hipMemcpy( dev_plys, moves->array, NUM_ROWS*NUM_ROWS * sizeof(Nimply), hipMemcpyHostToDevice ) );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(nim, NUM_ROWS, dev_results, dev_resultArray, dev_moves, dev_plys, dev_move);
        
        cudaHandleError( hipPeekAtLastError() );

        // Copy the move back from the GPU to the CPU
        cudaHandleError( hipMemcpy( move, dev_move, sizeof(Nimply), hipMemcpyDeviceToHost ) );

        // Free the memory allocated on the GPU
        hipFree( dev_results );
        hipFree( dev_resultArray );
        hipFree( dev_moves );
        hipFree( dev_plys );
        hipFree( dev_move );

        // Perform the move
        nim = nimming(nim, NUM_ROWS, move);
        player = 1 - player;

        printf("GPU Minmax - (%d, %d)\n", move->row, move->numSticks);
        printNim(nim, NUM_ROWS);
        printf("\n");

        // Free the memory we allocated on the CPU
        free(move);
        free(results->array);
        free(results);
        free(moves->array);
        free(moves);

        // The CPU perform a random move
        if (isNotEnded(nim)){
            nim = randomStrategy(nim, NUM_ROWS, true);
            player = 1 - player;
        }
    }

    printf(player == 0 ? "Minmax won!\n" : "Random won!\n");

    return 0;
}
