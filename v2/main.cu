#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nimlib.cuh"

#define NUM_ROWS 5

#define cudaHandleError(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    unsigned int maxMoves = NUM_ROWS * NUM_ROWS + 1;

    // Creation of the memory pointers
    unsigned char* moves; // the possible moves on the host CPU machine
    unsigned char* dev_moves; // the possible moves for the GPU device
    unsigned char* results;
    unsigned char* dev_results;

    // Initialize nim
    unsigned int nim = createNim(NUM_ROWS);
    printf("\n");
    printf("Initial board:\n");
    printNim(nim, NUM_ROWS);
    printf("\n");

    unsigned int player = 1;

    // Execute the minmax on the GPU device iteratively, until the game ends
    unsigned int a = 0;
    while(isNotEnded(nim) && a ==  0) {
        a++;
        // Allocate the memory on the CPU
        results = (unsigned char*)malloc(maxMoves * sizeof(unsigned char));
        moves = (unsigned char*)malloc(maxMoves * sizeof(unsigned char));
        if (!results || !moves) {
            fprintf(stderr, "malloc failure\n");
            exit(1);
        }
        // results[0] = 16;
        moves[0] = 16;

        // calculate the first level of the tree
        unsigned char numMoves = possibleMoves(nim, NUM_ROWS, moves, -1);

        // Allocate the memory on the GPU
        cudaHandleError( hipMalloc( (void**)&dev_moves, maxMoves * sizeof(unsigned char) ) );
        cudaHandleError( hipMalloc( (void**)&dev_results, maxMoves * sizeof(unsigned char) ) );

        // Copy nim to the GPU
        cudaHandleError( hipMemcpy( dev_moves, moves, maxMoves * sizeof(unsigned char), hipMemcpyHostToDevice ) );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(nim, NUM_ROWS, dev_moves, numMoves, dev_results);
        
        cudaHandleError( hipPeekAtLastError() );

        // Copy the move back from the GPU to the CPU
        cudaHandleError( hipMemcpy( results, dev_results, maxMoves * sizeof(unsigned char), hipMemcpyDeviceToHost ) );

        // Free the memory allocated on the GPU
        hipFree( dev_moves );
        hipFree( dev_results );

        // calculate the best move
        unsigned char move = minResultArray(results) & 127;

        // Perform the move
        nim = nimming(nim, NUM_ROWS, move);
        player = 1 - player;

        printf("GPU Minmax - (%d, %d)\n", (move >> 4) & 7, move & 15);
        printNim(nim, NUM_ROWS);
        printf("\n");

        // Free the memory we allocated on the CPU
        free(results);
        free(moves);

        // The CPU perform a random move
        if (isNotEnded(nim)){
            nim = randomStrategy(nim, NUM_ROWS, true);
            player = 1 - player;
        }
    }

    printf(player == 0 ? "Minmax won!\n" : "Random won!\n");

    return 0;
}
