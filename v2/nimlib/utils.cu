#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "utils.cuh"

__device__ void printResult(Result* result) {
    if (!result) {
        printf("Result - NULL\n");
        return;
    }
    printf("Result - Ply: (%d, %d), Val: %d\n", result->ply.row, result->ply.numSticks, result->val);
}

__device__ void printResultArray(ResultArray* resultArray, unsigned int level) {
    for (int i = 0; i < level; i++) {
        printf("   ");
    }
    if (!resultArray || !resultArray->array) {
        printf("ResultArray - NULL\n");
        return;
    }
    if (resultArray->numItems == 0) {
        printf("ResultArray - void\n");
        return;
    }
    printf("ResultArray - [\n");
    for (int i = 0; i < resultArray->numItems; i++) {
        printf("   ");
        for (int j = 0; j < level; j++) {
            printf("   ");
        }
        printResult(&(resultArray->array[i]));
    }
    for (int i = 0; i < level; i++) {
        printf("   ");
    }
    printf("]\n");
}
__device__ void printEntry(StackEntry* entry, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return;
    }
    if (!entry) {
        printf("StackEntry - NULL\n");
        return;
    }
    printf("StackEntry - {\n");
    printf("   ");
    printNim(entry->board, numRows);
    printf("   Alpha: %d, Beta: %d, Player: %d\n", entry->alpha, entry->beta, entry->player);
    printf("   Depth: %d, PlyIndex: %d, StackIndex: %d\n", entry->depth, entry->plyIndex, entry->stackIndex);
    printResultArray(&(entry->evaluations), 1);
    printf("   ");
    printResult(&(entry->result));
    printf("} \n");
}

__device__ void resultArrayPush(ResultArray* resultArray, unsigned int maxSize, Nimply* ply, int val) {
    if (resultArray->numItems == maxSize) {
        printf("the resultArray size is not enough!\n");
        return;
    }
    unsigned int index = resultArray->numItems;
    resultArray->array[index].ply = *ply;
    resultArray->array[index].val = val;
    resultArray->numItems++;
}

__device__ void minResultArray(ResultArray* results, Result* output) {
    if (results->numItems < 1) {
        printf("empty resultArray!\n");
        return;
    }
    unsigned int min_index = 0;
    int min_val = results->array[0].val;
    for (int i = 1; i < results->numItems; i++) {
        if ((results->array[i]).val < min_val) {
            min_index = i;
            min_val = (results->array[i]).val;
        }
    }
    output->ply = (results->array[min_index]).ply;
    output->val = min_val;
}

__device__ void maxResultArray(ResultArray* results, Result* output) {
    if (results->numItems < 1) {
        printf("empty ResultArray!\n");
        return;
    }
    unsigned int max_index = 0;
    int max_val = results->array[0].val;
    for (int i = 1; i < results->numItems; i++) {
        if ((results->array[i]).val > max_val) {
            max_index = i;
            max_val = (results->array[i]).val;
        }
    }
    output->ply = (results->array[max_index]).ply;
    output->val = max_val;
}

__device__ void stackPush(Stack* stack, unsigned int maxStackSize, unsigned int board, int alpha, int beta, int player, int depth, int plyIndex, int stackIndex, ResultArray* evaluations, Result* result) {
    if (stack->stackSize == maxStackSize) {
        printf("the stack size is not enough!\n");
        return;
    }
    unsigned int index = stack->stackSize;
    stack->array[index].board = board;
    stack->array[index].alpha = alpha;
    stack->array[index].beta = beta;
    stack->array[index].player = player;
    stack->array[index].depth = depth;
    stack->array[index].plyIndex = plyIndex;
    stack->array[index].stackIndex = stackIndex;
    if (evaluations) stack->array[index].evaluations = *evaluations;
    if (result) stack->array[index].result = *result;
    stack->stackSize++;
}

__device__ void stackPop(Stack* stack, StackEntry* entry) {
    if (stack->stackSize < 1) {
        printf("the stack is empty!\n");
        return;
    }
    stack->stackSize--;
    unsigned int index = stack->stackSize;
    entry->board = stack->array[index].board;
    entry->alpha = stack->array[index].alpha;
    entry->beta = stack->array[index].beta;
    entry->player = stack->array[index].player;
    entry->depth = stack->array[index].depth;
    entry->plyIndex = stack->array[index].plyIndex;
    entry->stackIndex = stack->array[index].stackIndex;
    entry->evaluations = stack->array[index].evaluations;
    entry->result = stack->array[index].result;
}
