#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nim.cuh"

__host__ __device__ void printNimply(Nimply* nimply) {
    if (!nimply) {
        printf("Nimply - NULL");
        return;
    }
    printf("Nimply - Row: %d, Num: %d\n", nimply->row, nimply->numSticks);
}

__host__ __device__ void printNim(unsigned int nim, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return;
    }
    printf("Nim - %d", nim & 1);
    unsigned int shift = 0;
    for (unsigned int i = 1; i < numRows; i++) {
        unsigned int mask;
        switch(i) {
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        shift += 4;
        printf(", %d", (nim >> shift) & mask);
    }
    printf("\n");
}

__host__ __device__ void printMovesArray(MovesArray* movesArray) {
    if (!movesArray || !movesArray->array) {
        printf("MovesArray - NULL\n");
        return;
    }
    if (movesArray->numItems == 0) {
        printf("MovesArray - void");
        return;
    }
    printf("MovesArray - [\n");
    for (int i = 0; i < movesArray->numItems; i++) {
        printf("   Row: %d, Num: %d\n", movesArray->array[i].row, movesArray->array[i].numSticks);
    }
    printf("]\n");
}

unsigned int createNim(unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return 0;
    }
    unsigned int nim = 0;
    unsigned int shift = 0;
    for (unsigned int i = 0; i < numRows; i++) {
        unsigned int num = i * 2 + 1;
        nim += (num << shift);
        shift += 4;
    }
    return nim;
}

__host__ __device__ bool isNotEnded(unsigned int nim) {
    return nim != 0;
}

__host__ __device__ unsigned int nimming(unsigned int nim, unsigned int numRows, Nimply* nimply) {
    unsigned int row = nimply->row;
    unsigned int numSticks = nimply->numSticks;
    if (numSticks < 1) {
        printf("Not a valid move!\n");
        return nim;
    }
    if (numRows <= row) {
        printf("Not enough rows!\n");
        return nim;
    }

    unsigned int mask;
    switch(row) {
        case 0:
            mask = 1;
            break;
        case 1:
            mask = 3;
            break;
        case 2:
            mask = 7;
            break;
        case 3:
            mask = 7;
            break;
        default:
            mask = 15;
    }
    unsigned int shift = 4 * row;

    unsigned int sticks = (nim >> shift) & mask;
    if (sticks < numSticks) {
        printf("Not enough sticks!\n");
        return nim;
    }
    unsigned int newMask = 4294967295 ^ (15 << shift);
    sticks = (sticks - numSticks) << shift;
    return (nim & newMask) | sticks;
}

__host__ __device__ void possibleMoves(unsigned int nim, unsigned int numRows, MovesArray* output) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return;
    }
    unsigned int shift = 0;
    unsigned int index = 0;
    for (unsigned int r = 0; r < numRows; r++) {
        unsigned int mask;
        switch(r) {
            case 0: // 1
                mask = 1;
                break;
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        unsigned int c = (nim >> shift) & mask;
        shift += 4;
        for (int o = 1; o <= c; o++) {
            output->array[index].row = r;
            output->array[index].numSticks = o;
            index++;
        }
    }
    output->numItems = index;
}

__host__ __device__ unsigned int nim_sum(unsigned int nim, unsigned int numRows) {
    if (numRows > 8) {
        printf("Not a valid number of rows!\n");
        return false;
    }
    unsigned int nim_sum = nim & 1;
    unsigned int shift = 0;
    for (unsigned int i = 1; i < numRows; i++) {
        unsigned int mask;
        switch(i) {
            case 1: // 3
                mask = 3;
                break;
            case 2: // 5
                mask = 7;
                break;
            case 3: // 7
                mask = 7;
                break;
            default: // > 7
                mask = 15;
        }
        shift += 4;
        nim_sum ^= ((nim >> shift) & mask);
    }
    return nim_sum; 
}
