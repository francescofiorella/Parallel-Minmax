#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nimlib.cuh"

__global__ void GPU_minmax(unsigned int nim, unsigned int numRows, ResultArray* results, Result* resultArray, MovesArray* moves, Nimply* plys, Nimply* ply) {
    // Associate arrays to classes
    results->array = resultArray;
    moves->array = plys;

    const unsigned int maxMoves = 25; 

    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned int stopComputation = 0;
    // stopComputation values:
    // 0 - Keep calculating
    // 1 - Calculate only the global result
    // 2 - Calculate the shared and the global result
    
    // if (bid == 0 && tid == 0) {
    //     // initialize the global output
    //     // the max number of results is equal to the available moves => the max num of moves is rows^2
        
    //     // calculate the first moves
    //     possibleMoves(nim, moves);
    //     results->numItems = moves->numItems;
    // }
    
    // __syncthreads();
    
    if (bid >= moves->numItems)
        return;
    
    __syncthreads();
    
    __shared__ MovesArray sharedMoves;
    __shared__ Nimply sharedPlys[maxMoves];
    sharedMoves.array = sharedPlys;
    
    __shared__ unsigned int sharedBoard;
    __shared__ int sharedPlayer;
    sharedPlayer = 1;
    if (tid == 0) {
        // calculate the new board and invert the current player
        // sharedBoard = nim;
        // select the move from bid
        // calculate the resulting board for the current move
        sharedBoard = nimming(nim, numRows, &(moves->array[bid]));
        sharedPlayer = 1 - sharedPlayer;

        // check if the game is ended, if yes update the results
        if (!isNotEnded(sharedBoard)) {
            Result res;
            res.ply = moves->array[bid];
            res.val = sharedPlayer;
            results->array[bid] = res;

            // jump to min/max ending evaluation if bid == 0 and tid == 0
            if (bid == 0)
                stopComputation = 1;
        }

        // calculate the new moves on shared array
        possibleMoves(sharedBoard, numRows, &sharedMoves);
    }

    __syncthreads();

    // works also if nim is ended
    if (stopComputation == 0 && tid >= sharedMoves.numItems)
        return;

    // __syncthreads();

    // declare Nim for this thread
    unsigned int newBoard;
    int player = sharedPlayer;
    __shared__ ResultArray sharedResults;
    __shared__ Result sharedResultArray[maxMoves];
    if (stopComputation == 0) {
        sharedResults.array = sharedResultArray;
        sharedResults.numItems = sharedMoves.numItems;

        // apply tid move
        newBoard = nimming(sharedBoard, numRows, &(sharedMoves.array[tid]));
        player = 1 - player;

        // check if nim is ended
        if (!isNotEnded(newBoard)) {
            Result res;
            res.ply = sharedMoves.array[tid];
            res.val = player;
            sharedResults.array[tid] = res;
            
            if (tid != 0)
                return;

            // stop the kernel only if tid != 0 else evaluate all the shared
            stopComputation = 2;
        }
    }

    __syncthreads();

    if (stopComputation == 0) {
        // start to calculate the minmax, store the result in sharedResults
        standard_minmax(newBoard, numRows, player, tid, sharedResults.array);

        if (tid != 0)
            return;
    }

    // when all secondary threads finished
    __syncthreads();

    if (stopComputation != 1) {
        // calculate the best move from the shared results
        Result sharedResult;
        maxResultArray(&sharedResults, &sharedResult);
        results->array[bid] = sharedResult;

        if (bid != 0)
            return;
    }

    __syncthreads();

    // calculate the best move from the global results
    Result lastResult;
    minResultArray(results, &lastResult);
    ply->row = lastResult.ply.row;
    ply->numSticks = lastResult.ply.numSticks;
}

// let's remove alpha beta pruning and max depth constrains
// push in the stack every move, using the same evaluations pointer
// at every move, push every move below
// use the depth value to discriminate between layers
// until 1024 (!?)

// modify the algorithm in order to perform one evaluation per thread, or
// use the same algorithm but run it several times in different threads



// bid e tid 0
// calcola mosse per nim originale
// inizializza vettore risultati [results]

// un bid per ogni mossa [25]

// tid 0
// applica mossa bid
// controlla se terminato -> se si, aggiungi a results
// inverti player
// calcola mosse per nuova board

// tid da 0 a 25
// applica mossa tid
// controlla se terminato -> se si, aggiungi a [?]
// inverti player
// fai partire loop per ogni thread

// sharedResults is the output
__device__ void standard_minmax(unsigned int nim, unsigned int numRows, int player, unsigned int tid, Result* sharedResults) {
    // printf("IN!\n");
    const unsigned int maxStackSize = 1000; /* TODO */
    const unsigned int maxMoves = 25; 

    // init the stack
    Stack stack;
    stack.stackSize = 0;
    StackEntry entries[maxStackSize];
    stack.array = entries;
    
    // push the very first empty entry
    stackPush(&stack, maxStackSize, 0, 0, 0, 0, 0, 0, 0, NULL, NULL);

    // push the first meaningful entry
    ResultArray evaluations;
    Result evaluationsArray[maxMoves];
    evaluations.array = evaluationsArray;
    evaluations.numItems = 0;
    stackPush(&stack, maxStackSize, nim, -1, 1, 1, 0, -1, stack.stackSize-1, &evaluations, NULL);

    StackEntry entry;
    
    // while there are moves to evaluate
    while (stack.stackSize > 1) {
        __syncthreads();

        stackPop(&stack, &entry);

        // stop if the max depth was reached
        if (entry.depth > 5) {
            Result res;
            // res.val = nim_sum(entry.board, numRows) == 0 ? entry.player : -entry.player;
            res.val = -entry.player;
            stack.array[entry.stackIndex].result = res;
            continue;
        }
        // stop if the game ended
        if (!isNotEnded(entry.board)) {
            Result res;
            res.val = entry.player;
            stack.array[entry.stackIndex].result = res;
            continue;
        }
        // calculate the posible moves
        MovesArray moves;
        Nimply plys[maxMoves];
        moves.array = plys;
        possibleMoves(entry.board, numRows, &moves);
        __syncthreads();
        // use the calculated result if it's not the first move
        if (entry.plyIndex != -1) {
            // exploit the previous result calculation
            int val = entry.result.val;
            resultArrayPush(&(entry.evaluations), maxMoves, &(moves.array[entry.plyIndex]), val);
            // update alpha or beta
            if (entry.player == 1) {
                if (entry.beta > val) entry.beta = val;
            } else {
                if (entry.alpha < val) entry.alpha = val;
            }
            // stop if it's the last move or it's time to prune
            if (entry.plyIndex == moves.numItems - 1 || entry.beta <= entry.alpha) {
                Result r;
                if (entry.player == 1) {
                    minResultArray(&(entry.evaluations), &r);
                } else {
                    maxResultArray(&(entry.evaluations), &r);
                }
                stack.array[entry.stackIndex].result = r;
                continue;
            }
        }
        __syncthreads();
        // evaluate the next move
        unsigned int newBoard;
        if (moves.array[entry.plyIndex+1].numSticks == 0) printNim(entry.board, numRows);
        newBoard = nimming(entry.board, numRows, &(moves.array[entry.plyIndex+1]));
        // push the previous state
        stackPush(&stack, maxStackSize, entry.board, entry.alpha, entry.beta, entry.player, entry.depth, entry.plyIndex + 1, entry.stackIndex, &(entry.evaluations), &(entry.result));
        __syncthreads();
        // push the current state (after making the move)
        ResultArray evaluations_;
        Result evaluationsArray_[maxMoves];
        evaluations_.array = evaluationsArray_;
        evaluations_.numItems = 0;
        stackPush(&stack, maxStackSize, newBoard, entry.alpha, entry.beta, -(entry.player), entry.depth + 1, -1, stack.stackSize - 1, &evaluations_, NULL);
    }
    stackPop(&stack, &entry);
    // printEntry(&entry);
    // push the result into the shared results
    sharedResults[tid] = entry.result;
}

unsigned int randomStrategy(unsigned int nim, unsigned int numRows, bool print) {
    MovesArray* moves;
    moves = (MovesArray*)malloc(sizeof(MovesArray));
    if (!moves) {
        fprintf(stderr, "malloc failure\n");
        exit(1);
    }

    unsigned int maxMoves = numRows * numRows;
    Nimply array[maxMoves];
    moves->array = array;
    possibleMoves(nim, numRows, moves);

    if (moves->numItems < 1) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % moves->numItems;
    Nimply* ply = &(moves->array[r]);
    nim = nimming(nim, numRows, ply);
    if (print){
        printf("Random - (%d, %d)\n", ply->row, ply->numSticks);
        printNim(nim, numRows);
        printf("\n");
    } 

    free(moves);
    return nim;
}
