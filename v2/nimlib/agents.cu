#include "hip/hip_runtime.h"
#include <cstdio>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "nimlib.cuh"

#define NUM_MOVES 26 // NUM_ROWS*NUM_ROWS + 1 (for the termination character)

__device__ __constant__ unsigned char dev_moves[NUM_MOVES]; // the possible moves for the GPU device

__global__ void GPU_minmax(unsigned int nim, unsigned int numRows, unsigned char numPlys, unsigned char* results) {
    // Associate thread id and block id
    unsigned int bid = blockIdx.x;
    unsigned int tid = threadIdx.x;

    unsigned int stopComputation = 0;
    // stopComputation values:
    // 0 - Keep calculating
    // 1 - Calculate only the global result
    // 2 - Calculate the shared and the global result
    
    if (bid >= numPlys)
        return;
    
    __syncthreads();
    
    __shared__ unsigned char sharedMoves[NUM_MOVES];
    __shared__ unsigned char sharedNumPlys;
    sharedMoves[0] = 16;
    sharedNumPlys = 0;

    __shared__ unsigned int sharedBoard;
    __shared__ int sharedPlayer;
    sharedPlayer = 1;
    if (tid == 0) {
        // calculate the new board and invert the current player
        // select the move from bid
        // calculate the resulting board for the current move
        sharedBoard = nimming(nim, numRows, dev_moves[bid]);
        sharedPlayer = -sharedPlayer;

        // check if the game is ended, if yes update the results
        if (!isNotEnded(sharedBoard)) {
            // 0 -> -1
            // 1 -> 1
            results[bid] = sharedPlayer == -1 ? 0 + (dev_moves[bid] & 127) : 128 + (dev_moves[bid] & 127); // 128 = 1 << 7

            // jump to min/max ending evaluation if bid == 0 and tid == 0
            if (bid == 0)
                stopComputation = 1;
        }

        // calculate the new moves on shared array
        sharedNumPlys = possibleMoves(sharedBoard, numRows, sharedMoves, -1);
    }

    __syncthreads();

    // works also if nim is ended
    if (stopComputation == 0 && tid >= sharedNumPlys)
        return;

    // declare Nim for this thread
    unsigned int newBoard;
    int player = sharedPlayer;
    __shared__ unsigned char sharedResults[NUM_MOVES];
    sharedResults[0] = 16;
    if (stopComputation == 0) {
        // apply tid move
        unsigned char move = sharedMoves[tid];
        newBoard = nimming(sharedBoard, numRows, move);
        player = -player;

        // check if nim is ended
        if (!isNotEnded(newBoard)) {
            sharedResults[tid] = player == -1 ? 0 + (move & 127) : 128 + (move & 127);
            
            if (tid != 0)
                return;

            // stop the kernel only if tid != 0 else evaluate all the shared
            stopComputation = 2;
        }
    }

    __syncthreads();

    if (stopComputation == 0) {
        // start to calculate the minmax, store the result in sharedResults
        standard_minmax(newBoard, numRows, player, tid, sharedResults);
    }

    if (tid != 0)
        return;

    // when all secondary threads finished
    __syncthreads();

    if (stopComputation != 1) {
        // calculate the best move from the shared results
        sharedResults[sharedNumPlys] = 16;
        results[bid] = maxResultArray(sharedResults);
    }

    if (bid != 0)
        return;

    // insert the termination char
    results[numPlys] = 16;
}

// sharedResults is the output
__device__ void standard_minmax(unsigned int nim, unsigned int numRows, int player, unsigned int tid, unsigned char* sharedResults) {
    const unsigned int maxDepth = 5;
    const unsigned int maxStackSize = 8;
    /*
    | Max Depth | Max Stack Size |
    | --------- | -------------- |
    | 1         | 4              |
    | 2         | 5              |
    | 3         | 6              |
    | 4         | 7              |
    | 5         | 8              |
    | 6         | 9              |
    | 7         | 10             |
    */

    // init the stack
    Stack stack;
    stack.stackSize = 0;
    StackEntry entries[maxStackSize];
    stack.array = entries;
    
    // push the very first empty entry
    stackPush(&stack, maxStackSize, 0, 0, 0, 0, 0, 0, 0, NULL, 16);

    // push the first meaningful entry
    unsigned char evaluations[maxStackSize-1][NUM_MOVES];
    evaluations[0][0] = 16;
    stackPush(&stack, maxStackSize, nim, -1, 1, 1, 0, -1, stack.stackSize-1, evaluations[0], 16);

    StackEntry entry;
    
    // while there are moves to evaluate
    while (stack.stackSize > 1) {
        __syncthreads();

        stackPop(&stack, &entry);

        // stop if the max depth was reached
        if (entry.depth > maxDepth) {
            stack.array[entry.stackIndex].result = entry.player == -1 ? 128 : 0;
            continue;
        }
        // stop if the game ended
        if (!isNotEnded(entry.board)) {
            stack.array[entry.stackIndex].result = entry.player == -1 ? 0 : 128;
            continue;
        }
        // calculate the posible moves
        unsigned char curr_move = possibleMoves(entry.board, numRows, NULL, entry.plyIndex+1);
        __syncthreads();
        // use the calculated result if it's not the first move
        if (entry.plyIndex != -1) {
            unsigned char prev_move = possibleMoves(entry.board, numRows, NULL, entry.plyIndex);
            // exploit the previous result calculation
            entry.evaluations[entry.plyIndex] = (entry.result & 128) + prev_move;
            entry.evaluations[entry.plyIndex + 1] = 16;
            int val = entry.result >> 7 == 0 ? -1 : 1;
            // update alpha or beta
            if (entry.player == 1) {
                if (entry.beta > val) entry.beta = val;
            } else {
                if (entry.alpha < val) entry.alpha = val;
            }
            // stop if it's the last move or it's time to prune
            if (curr_move == 16 || entry.beta <= entry.alpha) {
                unsigned char r;
                if (entry.player == 1) {
                    r = minResultArray(entry.evaluations);
                } else {
                    r = maxResultArray(entry.evaluations);
                }
                stack.array[entry.stackIndex].result = r;
                continue;
            }
        }
        __syncthreads();
        // evaluate the next move
        unsigned int newBoard;
        newBoard = nimming(entry.board, numRows, curr_move);
        // push the previous state
        stackPush(&stack, maxStackSize, entry.board, entry.alpha, entry.beta, entry.player, entry.depth, entry.plyIndex + 1, entry.stackIndex, entry.evaluations, entry.result);
        __syncthreads();
        // push the current state (after making the move)
        evaluations[stack.stackSize-1][0] = 16;
        stackPush(&stack, maxStackSize, newBoard, entry.alpha, entry.beta, -(entry.player), entry.depth + 1, -1, stack.stackSize - 1, evaluations[stack.stackSize-1], 16);
    }
    stackPop(&stack, &entry);
    // push the result into the shared results
    sharedResults[tid] = entry.result;
}

unsigned int randomStrategy(unsigned int nim, unsigned int numRows, bool print) {
    unsigned int maxMoves = numRows * numRows + 1;
    unsigned char moves[maxMoves];
    moves[0] = 16;
    unsigned char numMoves = possibleMoves(nim, numRows, moves, -1);

    if (numMoves == 0) {
        fprintf(stderr, "There are no moves available!\n");
        exit(1);
    }
    
    srand(time(NULL));
    int r = rand() % numMoves;
    unsigned char ply = moves[r];
    nim = nimming(nim, numRows, ply);
    if (print){
        printf("Random - (%d, %d)\n", (ply >> 4) & 7, ply & 15);
        printNim(nim, numRows);
        printf("\n");
    } 

    return nim;
}
