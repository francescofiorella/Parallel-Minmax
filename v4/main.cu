#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <stdlib.h>
#include <cmath>
#include <hip/hip_runtime.h>
#include "./nimlib/nimlib.cuh"

#define NUM_ROWS 5

#define cudaHandleError(ans) {gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUerror: %s\nCode: %d\nFile: %s\nLine: %d\n", hipGetErrorString(code), code, file, line);
        if (abort) exit(code);
    }
}

int main(void) {
    // Setup block size and max block count
    dim3 grid = dim3(NUM_ROWS*NUM_ROWS);
    dim3 thread = dim3(NUM_ROWS*NUM_ROWS);

    // Creation of the memory pointers
    unsigned char* ply;
    unsigned char* dev_ply;

    // Initialize nim
    unsigned int nim = createNim(NUM_ROWS);
    printf("\n");
    printf("Initial board:\n");
    printNim(nim, NUM_ROWS);
    printf("\n");

    unsigned int player = 1;

    // Execute the minmax on the GPU device iteratively, until the game ends
    unsigned int a = 0;
    while(isNotEnded(nim) && a ==  0) {
        a++;
        // Allocate the memory on the CPU
        ply = (unsigned char*)malloc(sizeof(unsigned char));
        if (!ply) {
            fprintf(stderr, "malloc failure\n");
            exit(1);
        }

        // Allocate the memory on the GPU
        cudaHandleError( hipMalloc( (void**)&dev_ply, sizeof(unsigned char) ) );

        // Execute the minmax on the GPU device
        GPU_minmax<<<grid, thread>>>(nim, NUM_ROWS, dev_ply);
        
        cudaHandleError( hipPeekAtLastError() );

        // Copy the move back from the GPU to the CPU
        cudaHandleError( hipMemcpy( ply, dev_ply, sizeof(unsigned char), hipMemcpyDeviceToHost ) );

        // Free the memory allocated on the GPU
        hipFree( dev_ply );

        // Perform the move
        nim = nimming(nim, NUM_ROWS, *ply);
        player = 1 - player;

        printf("GPU Minmax - (%d, %d)\n", (*ply >> 4) & 7, *ply & 15);
        printNim(nim, NUM_ROWS);
        printf("\n");

        // Free the memory we allocated on the CPU
        free(ply);

        // The CPU perform a random move
        if (isNotEnded(nim)){
            nim = randomStrategy(nim, NUM_ROWS, true);
            player = 1 - player;
        }
    }

    printf(player == 0 ? "Minmax won!\n" : "Random won!\n");

    return 0;
}
